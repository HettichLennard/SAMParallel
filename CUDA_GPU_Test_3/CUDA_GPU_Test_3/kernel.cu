#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>
#include <random>
#include <fstream>
#include <math.h>
#include <string>
#include <vector>

#include <errno.h>
#include <signal.h>
#include <thread>

#include "System.h"
#include "ParseUtils.h"
#include "Options.h"
#include "SimpSolver.h"

using namespace Minisat;
using namespace std;

__global__
void init_rand(hiprandState* state, int a, int b, int feature_count) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < feature_count) {
        hiprand_init(a + index, b + 2 * index, 0, &state[index]);
    }
}


__global__
void calculate_clause_values(int clause_count, short* current_clause_values, short* min_clause_values, int* clause_list, int lits_in_clause_limit, short* clause_list_sizes, char* optimized_variant) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < clause_count) {
        min_clause_values[index] = 1;
        current_clause_values[index] = 0;

        int lit_value;

        for (int i = 0; i < clause_list_sizes[index]; i++) {
            lit_value = clause_list[lits_in_clause_limit * index + i];
            if (lit_value < 0) {
                min_clause_values[index]--;
                lit_value = -lit_value - 1;
                current_clause_values[index] -= optimized_variant[lit_value];
            }
            else {
                lit_value = lit_value - 1;
                current_clause_values[index] += optimized_variant[lit_value];
            }
        }
    }
}


__global__
void reset_uncovered(int feature_count, int* uncovered) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count && index > 0) {
        uncovered[index - 1] = 0;
    }
}


__global__
void calculate_interactions(int parallel_limit, int feature_count, unsigned long long feature_interaction_count, int* current_sample_size, char* current_sample, int* skip_sample_index, char* feature_interactions, int* uncovered) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {

        int loop_index = 0;

        unsigned long long inter_index;

        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;
        char value_i, value_j;

        int lit_i_index, lit_j_index;

        double temp_i_value;

        while (loop_index * parallel_limit + index < feature_interaction_count) {

            inter_index = loop_index * parallel_limit + index;

            temp_i_value = 0.5 * (sqrt(double(inter_index) * 8.0 + 1.0) + 1.0);
            temp_i_value = floor(temp_i_value);

            lit_i_index = int(temp_i_value);

            lit_j_index = inter_index - ((lit_i_index - 1) * lit_i_index) / 2;

            inter_off_off = 1;
            inter_off_on = 1;
            inter_on_off = 1;
            inter_on_on = 1;

            for (int sample = 0; sample < current_sample_size[0]; sample++) {
                if (skip_sample_index[0] != sample) {
                    value_i = current_sample[lit_i_index + feature_count * sample];
                    value_j = current_sample[lit_j_index + feature_count * sample];

                    if (value_i == 0 && value_j == 0) inter_off_off = 0;
                    else if (value_i == 0 && value_j == 1) inter_off_on = 0;
                    else if (value_i == 1 && value_j == 0) inter_on_off = 0;
                    else if (value_i == 1 && value_j == 1) inter_on_on = 0;

                    if (inter_off_off == 0 && inter_off_on == 0 && inter_on_off == 0 && inter_on_on == 0) break;
                }
            }

            if (inter_off_off == 0 && inter_off_on == 0 && inter_on_off == 0 && inter_on_on == 0) {
                feature_interactions[inter_index] = 0;
            }
            else {
                inter_bin_encoding = inter_off_off + 2 * inter_off_on + 4 * inter_on_off + 8 * inter_on_on;

                feature_interactions[inter_index] = inter_bin_encoding;

                atomicAdd(&uncovered[lit_i_index - 1], inter_off_off + inter_off_on + inter_on_off + inter_on_on);
            }

            loop_index++;
        }
    }
}


__global__
void append_interactions(int parallel_limit, int feature_count, unsigned long long feature_interaction_count, char* optimized_variant, char* feature_interactions, int* uncovered) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {

        int loop_index = 0;

        unsigned long long inter_index;

        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;
        char value_i, value_j;

        int lit_i_index, lit_j_index;

        double temp_i_value;

        while (loop_index * parallel_limit + index < feature_interaction_count) {

            inter_index = loop_index * parallel_limit + index;

            inter_bin_encoding = feature_interactions[inter_index];

            if (inter_bin_encoding != 0) {

                if (inter_bin_encoding >= 8) {
                    inter_on_on = 1;
                    inter_bin_encoding -= 8;
                }
                else {
                    inter_on_on = 0;
                }

                if (inter_bin_encoding >= 4) {
                    inter_on_off = 1;
                    inter_bin_encoding -= 4;
                }
                else {
                    inter_on_off = 0;
                }

                if (inter_bin_encoding >= 2) {
                    inter_off_on = 1;
                    inter_bin_encoding -= 2;
                }
                else {
                    inter_off_on = 0;
                }

                if (inter_bin_encoding == 1) inter_off_off = 1;
                else inter_off_off = 0;

                temp_i_value = 0.5 * (sqrt(double(inter_index) * 8.0 + 1.0) + 1.0);
                temp_i_value = floor(temp_i_value);

                lit_i_index = int(temp_i_value);

                lit_j_index = inter_index - ((lit_i_index - 1) * lit_i_index) / 2;

                value_i = optimized_variant[lit_i_index];
                value_j = optimized_variant[lit_j_index];

                if (value_i == 0 && value_j == 0 && inter_off_off == 1) {
                    inter_off_off = 0;
                    atomicSub(&uncovered[lit_i_index - 1], 1);
                }
                else if (value_i == 0 && value_j == 1 && inter_off_on == 1) {
                    inter_off_on = 0;
                    atomicSub(&uncovered[lit_i_index - 1], 1);
                }
                else if (value_i == 1 && value_j == 0 && inter_on_off == 1) {
                    inter_on_off = 0;
                    atomicSub(&uncovered[lit_i_index - 1], 1);
                }
                else if (value_i == 1 && value_j == 1 && inter_on_on == 1) {
                    inter_on_on = 0;
                    atomicSub(&uncovered[lit_i_index - 1], 1);
                }

                if (inter_off_off == 0 && inter_off_on == 0 && inter_on_off == 0 && inter_on_on == 0) {
                    feature_interactions[inter_index] = 0;
                }
                else {
                    inter_bin_encoding = inter_off_off + 2 * inter_off_on + 4 * inter_on_off + 8 * inter_on_on;

                    feature_interactions[inter_index] = inter_bin_encoding;
                }
            }

            loop_index++;
        }
    }
}


__global__
void calculate_feature_probability(int feature_count, int* current_sample_size, char* current_sample, int* one_counts) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        char value;
        one_counts[index] = 0;

        for (int sample = 0; sample < current_sample_size[0]; sample++) {
            value = current_sample[index + feature_count * sample];

            if (value == 1) one_counts[index]++;
        }
    }
}


__global__
void init_sample(hiprandState* state, int feature_count, int* current_sample_size, int sampled_variants_size, char* sampled_variants, int* one_counts) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        int one_count = one_counts[index];
        float prob;
        float prob_threshold = float(one_count) / float(current_sample_size[0]);

        for (int sample = 0; sample < sampled_variants_size; sample++) {

            prob = hiprand_uniform(&(state[index]));

            if (prob < prob_threshold) sampled_variants[index + feature_count * sample] = 0;
            else sampled_variants[index + feature_count * sample] = 1;
        }
    }
}


__global__
void reset_novel_fi_counts(int sampled_variants_size, unsigned long long* novel_fi_counts) {
    for (int i = 0; i < sampled_variants_size; i++) {
        novel_fi_counts[i] = 0;
    }
}


__global__
void reset_novel_fi_count(unsigned long long* novel_fi_count) {
    novel_fi_count[0] = 0;
}


__global__
void calculate_sample_gain_all(int parallel_limit, int feature_count, unsigned long long feature_interaction_count, int sampled_variants_size, char* sampled_variants, unsigned long long* novel_fi_counts, char* feature_interactions) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {

        int loop_index = 0;

        unsigned long long inter_index;

        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;
        char value_i, value_j;

        int lit_i_index, lit_j_index;

        double temp_i_value;

        while (loop_index * parallel_limit + index < feature_interaction_count) {

            inter_index = loop_index * parallel_limit + index;

            inter_bin_encoding = feature_interactions[inter_index];

            if (inter_bin_encoding != 0) {
                if (inter_bin_encoding >= 8) {
                    inter_on_on = 1;
                    inter_bin_encoding -= 8;
                }
                else {
                    inter_on_on = 0;
                }

                if (inter_bin_encoding >= 4) {
                    inter_on_off = 1;
                    inter_bin_encoding -= 4;
                }
                else {
                    inter_on_off = 0;
                }

                if (inter_bin_encoding >= 2) {
                    inter_off_on = 1;
                    inter_bin_encoding -= 2;
                }
                else {
                    inter_off_on = 0;
                }

                if (inter_bin_encoding == 1) inter_off_off = 1;
                else inter_off_off = 0;

                temp_i_value = 0.5 * (sqrt(double(inter_index) * 8.0 + 1.0) + 1.0);
                temp_i_value = floor(temp_i_value);

                lit_i_index = int(temp_i_value);

                lit_j_index = inter_index - ((lit_i_index - 1) * lit_i_index) / 2;

                for (int sample = 0; sample < sampled_variants_size; sample++) {
                    value_i = sampled_variants[lit_i_index + feature_count * sample];
                    value_j = sampled_variants[lit_j_index + feature_count * sample];

                    if (value_i == 0 && value_j == 0 && inter_off_off == 1) atomicAdd(&novel_fi_counts[sample], 1);
                    else if (value_i == 0 && value_j == 1 && inter_off_on == 1) atomicAdd(&novel_fi_counts[sample], 1);
                    else if (value_i == 1 && value_j == 0 && inter_on_off == 1) atomicAdd(&novel_fi_counts[sample], 1);
                    else if (value_i == 1 && value_j == 1 && inter_on_on == 1) atomicAdd(&novel_fi_counts[sample], 1);
                }
            }

            loop_index++;
        }
    }
}


__global__
void calculate_sample_gain_optimized(int parallel_limit, int feature_count, unsigned long long feature_interaction_count, char* optimized_variant, unsigned long long* novel_fi_count, char* feature_interactions) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {

        int loop_index = 0;

        unsigned long long inter_index;

        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;
        char value_i, value_j;

        int lit_i_index, lit_j_index;

        double temp_i_value;

        while (loop_index * parallel_limit + index < feature_interaction_count) {

            inter_index = loop_index * parallel_limit + index;

            inter_bin_encoding = feature_interactions[inter_index];

            if (inter_bin_encoding != 0) {
                if (inter_bin_encoding >= 8) {
                    inter_on_on = 1;
                    inter_bin_encoding -= 8;
                }
                else {
                    inter_on_on = 0;
                }

                if (inter_bin_encoding >= 4) {
                    inter_on_off = 1;
                    inter_bin_encoding -= 4;
                }
                else {
                    inter_on_off = 0;
                }

                if (inter_bin_encoding >= 2) {
                    inter_off_on = 1;
                    inter_bin_encoding -= 2;
                }
                else {
                    inter_off_on = 0;
                }

                if (inter_bin_encoding == 1) inter_off_off = 1;
                else inter_off_off = 0;

                temp_i_value = 0.5 * (sqrt(double(inter_index) * 8.0 + 1.0) + 1.0);
                temp_i_value = floor(temp_i_value);

                lit_i_index = int(temp_i_value);

                lit_j_index = inter_index - ((lit_i_index - 1) * lit_i_index) / 2;

                value_i = optimized_variant[lit_i_index];
                value_j = optimized_variant[lit_j_index];

                if (value_i == 0 && value_j == 0 && inter_off_off == 1) atomicAdd(&novel_fi_count[0], 1);
                else if (value_i == 0 && value_j == 1 && inter_off_on == 1) atomicAdd(&novel_fi_count[0], 1);
                else if (value_i == 1 && value_j == 0 && inter_on_off == 1) atomicAdd(&novel_fi_count[0], 1);
                else if (value_i == 1 && value_j == 1 && inter_on_on == 1) atomicAdd(&novel_fi_count[0], 1);
            }

            loop_index++;
        }
    }
}


__global__
void find_max_sample(int sampled_variants_size, unsigned long long* novel_fi_counts, int* max_index) {
    max_index[0] = -1;
    unsigned long long temp_count, max_count = 0;

    for (int i = 0; i < sampled_variants_size; i++) {
        temp_count = novel_fi_counts[i];
        if (temp_count >= max_count) {
            max_count = temp_count;
            max_index[0] = i;
        }
    }
}


__global__
void write_max_sample_to_optimized(int feature_count, char* sampled_variants, char* optimized_variant, int* max_index) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        optimized_variant[index] = sampled_variants[feature_count * max_index[0] + index];
    }
}


__global__
void write_selected_sample_to_optimized(int feature_count, char* current_sample, char* optimized_variant, int* selected_index) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        optimized_variant[index] = current_sample[feature_count * selected_index[0] + index];
    }
}


__global__
void reset_M_values(int feature_count, int* M_values) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        M_values[index] = 0;
    }
}


__global__
void calculate_init_M_values(int parallel_limit, int feature_count, char* feature_interactions, char* optimized_variant, int* M_values) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {

        int loop_index = 0;

        unsigned long long inter_index;

        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;

        char value;

        int lit_i_index, lit_j_index;

        while (loop_index * parallel_limit + index < feature_count * feature_count) {

            inter_index = loop_index * parallel_limit + index;

            lit_j_index = inter_index % feature_count;
            lit_i_index = (inter_index - lit_j_index) / feature_count;

            if (lit_i_index != lit_j_index) {

                if (lit_j_index < lit_i_index) {
                    inter_index = ((lit_i_index - 1) * lit_i_index) / 2 + lit_j_index;
                }
                else if (lit_j_index > lit_i_index) {
                    inter_index = ((lit_j_index - 1) * lit_j_index) / 2 + lit_i_index;
                }

                inter_bin_encoding = feature_interactions[inter_index];

                if (inter_bin_encoding != 0) {
                    if (inter_bin_encoding >= 8) {
                        inter_on_on = 1;
                        inter_bin_encoding -= 8;
                    }
                    else {
                        inter_on_on = 0;
                    }

                    if (inter_bin_encoding >= 4) {
                        inter_on_off = 1;
                        inter_bin_encoding -= 4;
                    }
                    else {
                        inter_on_off = 0;
                    }

                    if (inter_bin_encoding >= 2) {
                        inter_off_on = 1;
                        inter_bin_encoding -= 2;
                    }
                    else {
                        inter_off_on = 0;
                    }

                    if (inter_bin_encoding == 1) inter_off_off = 1;
                    else inter_off_off = 0;

                    value = optimized_variant[lit_j_index];

                    if (lit_j_index < lit_i_index) {
                        if (inter_on_on == 1) atomicAdd(&M_values[lit_i_index], value);
                        if (inter_on_off == 1) atomicAdd(&M_values[lit_i_index], -value + 1);
                        if (inter_off_on == 1) atomicAdd(&M_values[lit_i_index], -value);
                        if (inter_off_off == 1) atomicAdd(&M_values[lit_i_index], value - 1);
                    }
                    else if (lit_j_index > lit_i_index) {
                        if (inter_on_on == 1) atomicAdd(&M_values[lit_i_index], value);
                        if (inter_on_off == 1) atomicAdd(&M_values[lit_i_index], -value);
                        if (inter_off_on == 1) atomicAdd(&M_values[lit_i_index], -value + 1);
                        if (inter_off_off == 1) atomicAdd(&M_values[lit_i_index], value - 1);
                    }
                }
            }

            loop_index++;
        }
    }
}


__global__
void reset_flip_counts(int feature_count, int* is_flip_valid) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        is_flip_valid[index] = 0;
    }
}


__global__
void reset_dual_flip_counts(int parallel_limit, int feature_count, unsigned long long feature_interaction_count, int* is_flip_valid, int* dual_invalids, int* dual_valids, int* valid_thresholds) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int loop_index = 0;

    if (index < feature_count) {
        is_flip_valid[index] = 0;
        // valid_thresholds[index] = 0;
    }

    /*
    if (index < parallel_limit) {
        while (loop_index * parallel_limit + index < feature_interaction_count) {
            dual_invalids[loop_index * parallel_limit + index] = 0;
            dual_valids[loop_index * parallel_limit + index] = 0;

            loop_index++;
        }
    }
    */
}


__global__
void calculate_flip_validity_optimized(int parallel_limit, int feature_count, unsigned long long total_lits, short* current_clause_values, short* min_clause_values, int* feat_to_clause_reference, int* clause_to_feat_reference, char* optimized_variant, int* is_flip_valid) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {

        int loop_index = 0;
        int clause_index, lit_index;
        bool is_negative;
        char value_i;

        while (loop_index * parallel_limit + index < total_lits) {

            clause_index = feat_to_clause_reference[loop_index * parallel_limit + index];
            lit_index = clause_to_feat_reference[loop_index * parallel_limit + index];

            value_i = optimized_variant[lit_index];

            if (clause_index < 0) {
                clause_index = -clause_index - 1;
                is_negative = true;
            }
            else {
                clause_index = clause_index - 1;
                is_negative = false;
            }

            if (current_clause_values[clause_index] == min_clause_values[clause_index]) {
                if (is_negative) {
                    if (value_i == 0) atomicAdd(&is_flip_valid[lit_index], 1);
                }
                else {
                    if (value_i == 1) atomicAdd(&is_flip_valid[lit_index], 1);
                }
            }

            loop_index++;
        }
    }
}


__global__
void calculate_dual_flip_validity_optimized(int parallel_limit, int feature_count, unsigned long long total_lits, short* current_clause_values, short* min_clause_values, int* feat_to_clause_reference, int* clause_to_feat_reference, int* clause_list, short* clause_list_sizes, int lits_in_clause_limit, char* optimized_variant, int* is_flip_valid, int* dual_invalids, int* dual_valids, int* valid_thresholds) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {

        int loop_index = 0;
        int clause_index, lit_i_index, lit_j_index, delta_value, delta_value_temp;
        bool is_i_negative, is_j_negative, is_invalid;
        char value_i, value_j;

        unsigned long long inter_index;

        while (loop_index * parallel_limit + index < total_lits) {

            clause_index = feat_to_clause_reference[loop_index * parallel_limit + index];
            lit_i_index = clause_to_feat_reference[loop_index * parallel_limit + index];

            value_i = optimized_variant[lit_i_index];

            if (clause_index < 0) {
                clause_index = -clause_index - 1;
                is_i_negative = true;
            }
            else {
                clause_index = clause_index - 1;
                is_i_negative = false;
            }

            if (is_i_negative) {
                if (value_i == 0) delta_value = -1;
                else delta_value = 1;
            }
            else {
                if (value_i == 1) delta_value = -1;
                else delta_value = 1;
            }

            if (current_clause_values[clause_index] + delta_value < min_clause_values[clause_index]) {
                // atomicAdd(&valid_thresholds[lit_i_index], 1);
                atomicAdd(&is_flip_valid[lit_i_index], 1);
                // is_invalid = true;
            }
            /*
            else {
                is_invalid = false;
            }
            */

            /*
            if (lit_i_index > 0) {

                for (int i = 0; i < clause_list_sizes[clause_index]; i++) {
                    lit_j_index = clause_list[clause_index * lits_in_clause_limit + i];

                    if (lit_j_index < 0) {
                        lit_j_index = -lit_j_index - 1;
                        is_j_negative = true;
                    }
                    else {
                        lit_j_index = lit_j_index - 1;
                        is_j_negative = false;
                    }

                    if (lit_j_index < lit_i_index) {
                        value_j = optimized_variant[lit_j_index];

                        if (is_j_negative) {
                            if (value_j == 0) delta_value_temp = delta_value - 1;
                            else delta_value_temp = delta_value + 1;
                        }
                        else {
                            if (value_j == 1) delta_value_temp = delta_value - 1;
                            else delta_value_temp = delta_value + 1;
                        }

                        if (!is_invalid) {
                            if (current_clause_values[clause_index] + delta_value_temp < min_clause_values[clause_index]) {
                                inter_index = ((lit_i_index - 1) * lit_i_index) / 2 + lit_j_index;
                                atomicAdd(&dual_invalids[inter_index], 1);
                            }
                        }
                        else {
                            if (current_clause_values[clause_index] + delta_value_temp >= min_clause_values[clause_index]) {
                                inter_index = ((lit_i_index - 1) * lit_i_index) / 2 + lit_j_index;
                                atomicAdd(&dual_valids[inter_index], 1);
                            }
                        }
                    }
                }
            }
            */

            loop_index++;

        }
    }
}


__global__
void calculate_max_gain_dual_with_validity_check(int parallel_limit, int feature_count, unsigned long long feature_interaction_count, char* feature_interactions, char* optimized_variant, int* M_values, int* max_indices_i_m, int* max_indices_j_m, int* max_gains_m, int step_size, int* current_step, int* is_flip_valid, int* dual_invalids, int* dual_valids, int* valid_thresholds, char* are_feature_pairs_dependent) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int loop_index = 0;

    if (index < parallel_limit) {
        unsigned long long inter_index;

        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;

        int M_value_i, M_value_j, M_value_i_reduced, M_value_j_reduced, gain_value, q_value;

        char value_i, value_j;

        int lit_i_index, lit_j_index;

        double temp_i_value;

        bool valid;

        max_indices_i_m[index] = -1;
        max_indices_j_m[index] = -1;
        max_gains_m[index] = 0;

        while (loop_index * parallel_limit + index < feature_interaction_count) {

            inter_index = loop_index * parallel_limit + index;

            temp_i_value = 0.5 * (sqrt(double(inter_index) * 8.0 + 1.0) + 1.0);
            temp_i_value = floor(temp_i_value);

            lit_i_index = int(temp_i_value);

            lit_j_index = inter_index - ((lit_i_index - 1) * lit_i_index) / 2;

            if (lit_j_index % step_size == current_step[0]) {

                value_i = optimized_variant[lit_i_index];
                M_value_i = M_values[lit_i_index];

                value_j = optimized_variant[lit_j_index];
                M_value_j = M_values[lit_j_index];

                if (value_j == 0) {
                    if (is_flip_valid[lit_i_index] == 0) {
                        gain_value = M_value_i * (1 - 2 * value_i);

                        if (gain_value > max_gains_m[index]) {
                            max_gains_m[index] = gain_value;
                            max_indices_i_m[index] = lit_i_index;
                            max_indices_j_m[index] = -1;
                        }
                    }
                }

                valid = false;

                if (are_feature_pairs_dependent[inter_index] == 0) {
                    if (is_flip_valid[lit_i_index] == 0 && is_flip_valid[lit_j_index] == 0) valid = true;
                }
                /*
                else {
                    if (dual_invalids[inter_index] == 0) valid = true;

                    if (valid_thresholds[index] > 0) {
                        if (dual_valids[inter_index] < valid_thresholds[index]) valid = false;
                    }
                }
                */

                if (valid) {

                    inter_bin_encoding = feature_interactions[inter_index];

                    if (inter_bin_encoding != 0) {

                        if (inter_bin_encoding >= 8) {
                            inter_on_on = 1;
                            inter_bin_encoding -= 8;
                        }
                        else {
                            inter_on_on = 0;
                        }

                        if (inter_bin_encoding >= 4) {
                            inter_on_off = 1;
                            inter_bin_encoding -= 4;
                        }
                        else {
                            inter_on_off = 0;
                        }

                        if (inter_bin_encoding >= 2) {
                            inter_off_on = 1;
                            inter_bin_encoding -= 2;
                        }
                        else {
                            inter_off_on = 0;
                        }

                        if (inter_bin_encoding == 1) inter_off_off = 1;
                        else inter_off_off = 0;

                        q_value = 0;

                        if (inter_on_on == 1) q_value += 1;
                        if (inter_on_off == 1) q_value += -1;
                        if (inter_off_on == 1) q_value += -1;
                        if (inter_off_off == 1) q_value += 1;

                        M_value_i_reduced = M_value_i - q_value * value_j;
                        M_value_j_reduced = M_value_j - q_value * value_i;

                        gain_value = M_value_i_reduced * (1 - 2 * value_i) + M_value_j_reduced * (1 - 2 * value_j);
                        if (q_value != 0) gain_value += q_value * (-value_i * value_j + (1 - value_i) * (1 - value_j));

                    }
                    else {
                        gain_value = M_value_i * (1 - 2 * value_i) + M_value_j * (1 - 2 * value_j);
                    }

                    if (gain_value > max_gains_m[index]) {
                        max_gains_m[index] = gain_value;
                        max_indices_i_m[index] = lit_i_index;
                        max_indices_j_m[index] = lit_j_index;
                    }
                }

            }

            loop_index++;
        }
    }
}


__global__
void calculate_max_gain_dual_with_validity_check_2(int feature_count, char* feature_interactions, char* optimized_variant, int* M_values, int* max_indices_j_m, int* max_gains_m, int step_size, int* current_step, int* is_flip_valid, int* dual_invalids, int* dual_valids, int* valid_thresholds, char* are_feature_pairs_dependent) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {

        unsigned long long inter_index;
        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;

        int M_value_i, M_value_j, M_value_i_reduced, M_value_j_reduced, gain_value, q_value;
        char value_i = optimized_variant[index], value_j;
        int lit_value;

        max_indices_j_m[index] = -1;
        max_gains_m[index] = 0;

        M_value_i = M_values[index];

        if (is_flip_valid[index] == 0) {
            gain_value = M_value_i * (1 - 2 * value_i);

            if (gain_value > max_gains_m[index]) {
                max_gains_m[index] = gain_value;
                max_indices_j_m[index] = -1;
            }

            for (int j = current_step[0]; j < index; j += step_size) {

                if (is_flip_valid[j] == 0) {
                    value_j = optimized_variant[j];

                    M_value_j = M_values[j];

                    inter_index = ((index - 1) * index) / 2 + j;

                    if (are_feature_pairs_dependent[inter_index] == 0) {

                        inter_bin_encoding = feature_interactions[inter_index];

                        if (inter_bin_encoding != 0) {

                            if (inter_bin_encoding >= 8) {
                                inter_on_on = 1;
                                inter_bin_encoding -= 8;
                            }
                            else {
                                inter_on_on = 0;
                            }

                            if (inter_bin_encoding >= 4) {
                                inter_on_off = 1;
                                inter_bin_encoding -= 4;
                            }
                            else {
                                inter_on_off = 0;
                            }

                            if (inter_bin_encoding >= 2) {
                                inter_off_on = 1;
                                inter_bin_encoding -= 2;
                            }
                            else {
                                inter_off_on = 0;
                            }

                            if (inter_bin_encoding == 1) inter_off_off = 1;
                            else inter_off_off = 0;

                            q_value = 0;

                            if (inter_on_on == 1) q_value += 1;
                            if (inter_on_off == 1) q_value += -1;
                            if (inter_off_on == 1) q_value += -1;
                            if (inter_off_off == 1) q_value += 1;

                            M_value_i_reduced = M_value_i - q_value * value_j;
                            M_value_j_reduced = M_value_j - q_value * value_i;

                            gain_value = M_value_i_reduced * (1 - 2 * value_i) + M_value_j_reduced * (1 - 2 * value_j);
                            if (q_value != 0) gain_value += q_value * (-value_i * value_j + (1 - value_i) * (1 - value_j));

                        }
                        else {
                            gain_value = M_value_i * (1 - 2 * value_i) + M_value_j * (1 - 2 * value_j);

                        }

                        if (gain_value > max_gains_m[index]) {
                            max_gains_m[index] = gain_value;
                            max_indices_j_m[index] = j;
                        }
                    }
                }
            }
        }
    }
}


__global__
void preprocess_maximum(int parallel_limit, int batch_size, int* max_indices_i_m, int* max_indices_j_m, int* max_gains_m, int* max_indices_i_reduced, int* max_indices_j_reduced, int* max_gains_reduced) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < parallel_limit) {
        max_gains_reduced[index] = 0;
        max_indices_i_reduced[index] = -1;
        max_indices_j_reduced[index] = -1;

        int iterator;

        for (int i = 0; i < batch_size; i++) {
            iterator = index * batch_size + i;

            if (max_gains_m[iterator] > max_gains_reduced[index]) {
                max_gains_reduced[index] = max_gains_m[iterator];
                max_indices_i_reduced[index] = max_indices_i_m[iterator];
                max_indices_j_reduced[index] = max_indices_j_m[iterator];
            }
        }
    }
}


__global__
void adapt_M_values(int feature_count, char* feature_interactions, char* optimized_variant, int* M_values, int* max_index_m) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        unsigned long long inter_index;
        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;

        char value;

        int q_value = 0;

        if (max_index_m[0] != index) {

            if (max_index_m[0] < index) {
                inter_index = ((index - 1) * index) / 2 + max_index_m[0];
            }
            else if (max_index_m[0] > index) {
                inter_index = ((max_index_m[0] - 1) * max_index_m[0]) / 2 + index;
            }

            inter_bin_encoding = feature_interactions[inter_index];

            if (inter_bin_encoding != 0) {
                if (inter_bin_encoding >= 8) {
                    inter_on_on = 1;
                    inter_bin_encoding -= 8;
                }
                else {
                    inter_on_on = 0;
                }

                if (inter_bin_encoding >= 4) {
                    inter_on_off = 1;
                    inter_bin_encoding -= 4;
                }
                else {
                    inter_on_off = 0;
                }

                if (inter_bin_encoding >= 2) {
                    inter_off_on = 1;
                    inter_bin_encoding -= 2;
                }
                else {
                    inter_off_on = 0;
                }

                if (inter_bin_encoding == 1) inter_off_off = 1;
                else inter_off_off = 0;

                if (inter_on_on == 1) q_value += 1;
                if (inter_on_off == 1) q_value += -1;
                if (inter_off_on == 1) q_value += -1;
                if (inter_off_off == 1) q_value += 1;

                if (q_value != 0) {
                    value = optimized_variant[max_index_m[0]];
                    M_values[index] += q_value * (1 - 2 * value);
                }
            }
        }
    }
}


__global__
void adapt_M_values_dual(int feature_count, char* feature_interactions, char* optimized_variant, int* M_values, int* max_index_i_m, int* max_index_j_m) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        unsigned long long inter_index;
        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char inter_bin_encoding;

        char value_i = optimized_variant[max_index_i_m[0]];
        char value_j = optimized_variant[max_index_j_m[0]];

        int q_value = 0;

        if (max_index_i_m[0] != index) {

            if (max_index_i_m[0] < index) {
                inter_index = ((index - 1) * index) / 2 + max_index_i_m[0];
            }
            else if (max_index_i_m[0] > index) {
                inter_index = ((max_index_i_m[0] - 1) * max_index_i_m[0]) / 2 + index;
            }

            inter_bin_encoding = feature_interactions[inter_index];

            if (inter_bin_encoding != 0) {
                if (inter_bin_encoding >= 8) {
                    inter_on_on = 1;
                    inter_bin_encoding -= 8;
                }
                else {
                    inter_on_on = 0;
                }

                if (inter_bin_encoding >= 4) {
                    inter_on_off = 1;
                    inter_bin_encoding -= 4;
                }
                else {
                    inter_on_off = 0;
                }

                if (inter_bin_encoding >= 2) {
                    inter_off_on = 1;
                    inter_bin_encoding -= 2;
                }
                else {
                    inter_off_on = 0;
                }

                if (inter_bin_encoding == 1) inter_off_off = 1;
                else inter_off_off = 0;

                if (inter_on_on == 1) q_value += 1;
                if (inter_on_off == 1) q_value += -1;
                if (inter_off_on == 1) q_value += -1;
                if (inter_off_off == 1) q_value += 1;

                if (q_value != 0) {
                    M_values[index] += q_value * (1 - 2 * value_i);
                }
            }
        }

        if (max_index_j_m[0] != -1) {
            q_value = 0;

            if (max_index_j_m[0] != index) {

                if (max_index_j_m[0] < index) {
                    inter_index = ((index - 1) * index) / 2 + max_index_j_m[0];
                }
                else if (max_index_j_m[0] > index) {
                    inter_index = ((max_index_j_m[0] - 1) * max_index_j_m[0]) / 2 + index;
                }

                inter_bin_encoding = feature_interactions[inter_index];

                if (inter_bin_encoding != 0) {
                    if (inter_bin_encoding >= 8) {
                        inter_on_on = 1;
                        inter_bin_encoding -= 8;
                    }
                    else {
                        inter_on_on = 0;
                    }

                    if (inter_bin_encoding >= 4) {
                        inter_on_off = 1;
                        inter_bin_encoding -= 4;
                    }
                    else {
                        inter_on_off = 0;
                    }

                    if (inter_bin_encoding >= 2) {
                        inter_off_on = 1;
                        inter_bin_encoding -= 2;
                    }
                    else {
                        inter_off_on = 0;
                    }

                    if (inter_bin_encoding == 1) inter_off_off = 1;
                    else inter_off_off = 0;

                    if (inter_on_on == 1) q_value += 1;
                    if (inter_on_off == 1) q_value += -1;
                    if (inter_off_on == 1) q_value += -1;
                    if (inter_off_off == 1) q_value += 1;

                    if (q_value != 0) {
                        M_values[index] += q_value * (1 - 2 * value_j);
                    }
                }
            }
        }
    }
}


__global__
void adapt_clause_values(short* current_clause_values, int* clauses_with_lit, int* feat_to_clause_reference, unsigned long long* reference_start_points, char* optimized_variant, int* max_index_m) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < clauses_with_lit[max_index_m[0]]) {

        int clause_index = feat_to_clause_reference[reference_start_points[max_index_m[0]] + index];
        if (clause_index < 0) {
            clause_index = -clause_index - 1;

            if (optimized_variant[max_index_m[0]] == 0) {
                current_clause_values[clause_index] -= 1;
            }
            else {
                current_clause_values[clause_index] += 1;
            }
        }
        else {
            clause_index = clause_index - 1;

            if (optimized_variant[max_index_m[0]] == 0) {
                current_clause_values[clause_index] += 1;
            }
            else {
                current_clause_values[clause_index] -= 1;
            }
        }
    }
}


__global__
void optimize_variant(char* optimized_variant, int* max_index_m) {
    optimized_variant[max_index_m[0]] = 1 - optimized_variant[max_index_m[0]];
}


__global__
void optimize_variant_dual(char* optimized_variant, int* max_index_i_m, int* max_index_j_m) {
    if (max_index_j_m[0] == -1) {
        optimized_variant[max_index_i_m[0]] = 1 - optimized_variant[max_index_i_m[0]];
    }
    else {
        optimized_variant[max_index_i_m[0]] = 1 - optimized_variant[max_index_i_m[0]];
        optimized_variant[max_index_j_m[0]] = 1 - optimized_variant[max_index_j_m[0]];
    }
}


__global__
void write_optimized_to_current_sample(int feature_count, char* optimized_variant, int* insertion_point, char* current_sample) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < feature_count) {
        current_sample[feature_count * insertion_point[0] + index] = optimized_variant[index];
    }
}


__global__
void increase_current_sample_size(int* current_sample_size) {
    current_sample_size[0] += 1;
}


int main(void)
{
    setUsageHelp("USAGE: %s [options] <input-file> <result-output-file>\n\n  where input may be either in plain or gzipped DIMACS.\n");

    IntOption    verb("MAIN", "verb", "Verbosity level (0=silent, 1=some, 2=more).", 0, IntRange(0, 2));
    BoolOption   pre("MAIN", "pre", "Completely turn on/off any preprocessing.", false);
    StringOption dimacs("MAIN", "dimacs", "If given, stop after preprocessing and write the result to this file.");
    IntOption    cpu_lim("MAIN", "cpu-lim", "Limit on CPU time allowed in seconds.\n", INT32_MAX, IntRange(0, INT32_MAX));
    IntOption    mem_lim("MAIN", "mem-lim", "Limit on memory usage in megabytes.\n", INT32_MAX, IntRange(0, INT32_MAX));

    std::mt19937_64 rng;
    uint64_t timeSeed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    std::seed_seq ss{ uint32_t(timeSeed & 0xffffffff), uint32_t(timeSeed >> 32) };
    rng.seed(ss);
    std::uniform_real_distribution<double> unif(0, 1);
    std::uniform_real_distribution<double> unif2(0, 10000);
    std::uniform_real_distribution<double> unif3(0, 10000);

    vector<int> cnf_command_params;
    vec<Lit> lits;

    char* active_literals = nullptr, * inactive_literals = nullptr;
    int* clause_list = nullptr, * clause_list_c;
    short* clause_list_sizes = nullptr, * clause_list_sizes_c;
    short* current_clause_values_c, * min_clause_values_c;

    int* feat_to_clause_reference = nullptr, * feat_to_clause_reference_c;
    int* clause_to_feat_reference = nullptr, * clause_to_feat_reference_c;
    int* clauses_with_lit = nullptr, * clauses_with_lit_c, * current_reference_iterators = nullptr;
    unsigned long long* reference_start_points = nullptr, * reference_start_points_c;

    int* dual_invalids_c, int* dual_valids_c;
    int* valid_thresholds_c;
    char* are_feature_pairs_dependent, * are_feature_pairs_dependent_c;

    char* invalid_feature_interactions;
    int* invalids_uncovered;

    // ---------------------------
    const string cnf_file_name = "C:\\Users\\lenna\\Downloads\\bmc\\automotive.cnf";
    const bool validity_checking = true;
    const int lits_in_clause_limit = 25;
    const int sampled_variants_size = 100;
    // ---------------------------

    ifstream cnf_file(cnf_file_name);

    int feature_count = -1, init_clause_count, reduced_clause_count = 0;
    int feature_interaction_count; 

    string file_line;
    int ascii_value, lit_value, char_counter;
    bool is_lit_negative, is_comment, is_command;

    int max_lits_per_clause = 0;
    unsigned long long total_lits_in_file = 0;

    SimpSolver* S = new SimpSolver[sampled_variants_size];

    for (int i = 0; i < sampled_variants_size; i++) {
        S[i].eliminate(true);
        S[i].verbosity = verb;
        S[i].setRandomSeed(unif3(rng));
    }

    auto start0 = std::chrono::high_resolution_clock::now();

    if (cnf_file.is_open()) {
        while (getline(cnf_file, file_line)) {

            if (validity_checking) {
                if (feature_count != -1) {
                    for (int i = 0; i < feature_count; i++) {
                        inactive_literals[i] = 0;
                        active_literals[i] = 0;
                    }
                }
            }

            lit_value = 0;

            lits.clear();

            is_lit_negative = false;
            is_comment = false;
            is_command = false;

            char_counter = 0;

            for (char& c : file_line) {
                ascii_value = int(c);
                if (char_counter == 0 && ascii_value == 99) {
                    is_comment = true;
                    break;
                }

                if (char_counter == 0 && ascii_value == 112) is_command = true;

                if (ascii_value >= 48 && ascii_value <= 57) {
                    if (lit_value == 0) {
                        lit_value = ascii_value - 48;
                    }
                    else {
                        lit_value *= 10;
                        lit_value += ascii_value - 48;
                    }
                }
                else if (ascii_value == 45) {
                    is_lit_negative = true;
                }
                else {
                    if (!is_command) {
                        if (lit_value > 0) {
                            if (validity_checking) {
                                if (is_lit_negative) inactive_literals[lit_value - 1] = 1;
                                else active_literals[lit_value - 1] = 1;
                            }
                            else {
                                if (is_lit_negative) {
                                    lits.push(~mkLit(lit_value - 1));
                                    clause_list[reduced_clause_count * lits_in_clause_limit + clause_list_sizes[reduced_clause_count]] = -lit_value;
                                    clause_list_sizes[reduced_clause_count] += 1;
                                }
                                else {
                                    lits.push(mkLit(lit_value - 1));
                                    clause_list[reduced_clause_count * lits_in_clause_limit + clause_list_sizes[reduced_clause_count]] = lit_value;
                                    clause_list_sizes[reduced_clause_count] += 1;
                                }
                            }
                        }
                    }
                    else {
                        if (lit_value > 0) {
                            cnf_command_params.push_back(lit_value);
                        }
                    }

                    lit_value = 0;
                    is_lit_negative = false;
                }

                char_counter++;
            }

            if (is_comment) continue;

            if (!is_command) {
                if (lit_value > 0) {
                    if (validity_checking) {
                        if (is_lit_negative) inactive_literals[lit_value - 1] = 1;
                        else active_literals[lit_value - 1] = 1;
                    }
                    else {
                        if (is_lit_negative) {
                            lits.push(~mkLit(lit_value - 1));
                            clause_list[reduced_clause_count * lits_in_clause_limit + clause_list_sizes[reduced_clause_count]] = -lit_value;
                            clause_list_sizes[reduced_clause_count] += 1;
                        }
                        else {
                            lits.push(mkLit(lit_value - 1));
                            clause_list[reduced_clause_count * lits_in_clause_limit + clause_list_sizes[reduced_clause_count]] = lit_value;
                            clause_list_sizes[reduced_clause_count] += 1;
                        }
                    }
                }

                bool irrelevant_clause = false;

                if (validity_checking) {
                    for (int n = 0; n < feature_count; n++) {
                        if (active_literals[n] == 1 && inactive_literals[n] == 1) {
                            irrelevant_clause = true;
                            break;
                        }
                    }
                }

                if (!irrelevant_clause) {
                    if (validity_checking) {
                        for (int n = 0; n < feature_count; n++) {
                            if (inactive_literals[n] == 1) {
                                lits.push(~mkLit(n));
                                clause_list[reduced_clause_count * lits_in_clause_limit + clause_list_sizes[reduced_clause_count]] = -(n + 1);
                                clause_list_sizes[reduced_clause_count] += 1;
                            }
                            else if (active_literals[n] == 1) {
                                lits.push(mkLit(n));
                                clause_list[reduced_clause_count * lits_in_clause_limit + clause_list_sizes[reduced_clause_count]] = (n + 1);
                                clause_list_sizes[reduced_clause_count] += 1;
                            }
                        }
                    }

                    if (clause_list_sizes[reduced_clause_count] == 1) {
                        int lit1;
                        bool is_1_negative;
                        unsigned long long inter_index;
                        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
                        char inter_bin_encoding;
                        int old_invalids_count, new_invalids_count;

                        lit1 = clause_list[reduced_clause_count * lits_in_clause_limit];

                        if (lit1 < 0) {
                            lit1 = -lit1 - 1;
                            is_1_negative = true;
                        }
                        else {
                            lit1 = lit1 - 1;
                            is_1_negative = false;
                        }

                        for (int lit2 = 0; lit2 < feature_count; lit2++) {
                            if (lit1 != lit2) {

                                if (lit1 < lit2) inter_index = ((lit2 - 1) * lit2) / 2 + lit1;
                                else inter_index = ((lit1 - 1) * lit1) / 2 + lit2;

                                inter_bin_encoding = invalid_feature_interactions[inter_index];

                                if (inter_bin_encoding != 0) {
                                    if (inter_bin_encoding >= 8) {
                                        inter_on_on = 1;
                                        inter_bin_encoding -= 8;
                                    }
                                    else {
                                        inter_on_on = 0;
                                    }

                                    if (inter_bin_encoding >= 4) {
                                        inter_on_off = 1;
                                        inter_bin_encoding -= 4;
                                    }
                                    else {
                                        inter_on_off = 0;
                                    }

                                    if (inter_bin_encoding >= 2) {
                                        inter_off_on = 1;
                                        inter_bin_encoding -= 2;
                                    }
                                    else {
                                        inter_off_on = 0;
                                    }

                                    if (inter_bin_encoding == 1) inter_off_off = 1;
                                    else inter_off_off = 0;

                                    old_invalids_count = inter_on_on + inter_on_off + inter_off_on + inter_off_off;

                                    if (lit1 < lit2) {
                                        if (is_1_negative) {
                                            inter_on_on = 1;
                                            inter_off_on = 1;
                                        }
                                        else {
                                            inter_on_off = 1;
                                            inter_off_off = 1;
                                        }
                                    }
                                    else {
                                        if (is_1_negative) {
                                            inter_on_on = 1;
                                            inter_on_off = 1;
                                        }
                                        else {
                                            inter_off_on = 1;
                                            inter_off_off = 1;
                                        }
                                    }

                                    new_invalids_count = inter_on_on + inter_on_off + inter_off_on + inter_off_off;

                                    if (lit1 < lit2) invalids_uncovered[lit2 - 1] += new_invalids_count - old_invalids_count;
                                    else invalids_uncovered[lit1 - 1] += new_invalids_count - old_invalids_count;

                                    inter_bin_encoding = inter_off_off + 2 * inter_off_on + 4 * inter_on_off + 8 * inter_on_on;

                                    invalid_feature_interactions[inter_index] = inter_bin_encoding;
                                }
                                else {
                                    inter_on_on = 0;
                                    inter_on_off = 0;
                                    inter_off_on = 0;
                                    inter_off_off = 0;

                                    if (lit1 < lit2) {
                                        if (is_1_negative) {
                                            inter_on_on = 1;
                                            inter_off_on = 1;
                                        }
                                        else {
                                            inter_on_off = 1;
                                            inter_off_off = 1;
                                        }
                                    }
                                    else {
                                        if (is_1_negative) {
                                            inter_on_on = 1;
                                            inter_on_off = 1;
                                        }
                                        else {
                                            inter_off_on = 1;
                                            inter_off_off = 1;
                                        }
                                    }

                                    if (lit1 < lit2) invalids_uncovered[lit2 - 1] += 2;
                                    else invalids_uncovered[lit1 - 1] += 2;

                                    inter_bin_encoding = inter_off_off + 2 * inter_off_on + 4 * inter_on_off + 8 * inter_on_on;

                                    invalid_feature_interactions[inter_index] = inter_bin_encoding;
                                }
                            }
                        }
                    }
                    else if (clause_list_sizes[reduced_clause_count] == 2) {
                        int lit1, lit2;
                        bool is_1_negative, is_2_negative;
                        unsigned long long inter_index;
                        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
                        char inter_bin_encoding;
                        int old_invalids_count, new_invalids_count;

                        lit1 = clause_list[reduced_clause_count * lits_in_clause_limit];
                        lit2 = clause_list[reduced_clause_count * lits_in_clause_limit + 1];

                        if (lit1 < 0) {
                            lit1 = -lit1 - 1;
                            is_1_negative = true;
                        }
                        else {
                            lit1 = lit1 - 1;
                            is_1_negative = false;
                        }

                        if (lit2 < 0) {
                            lit2 = -lit2 - 1;
                            is_2_negative = true;
                        }
                        else {
                            lit2 = lit2 - 1;
                            is_2_negative = false;
                        }

                        if (lit1 < lit2) inter_index = ((lit2 - 1) * lit2) / 2 + lit1;
                        else inter_index = ((lit1 - 1) * lit1) / 2 + lit2;

                        inter_bin_encoding = invalid_feature_interactions[inter_index];

                        if (inter_bin_encoding != 0) {
                            if (inter_bin_encoding >= 8) {
                                inter_on_on = 1;
                                inter_bin_encoding -= 8;
                            }
                            else {
                                inter_on_on = 0;
                            }

                            if (inter_bin_encoding >= 4) {
                                inter_on_off = 1;
                                inter_bin_encoding -= 4;
                            }
                            else {
                                inter_on_off = 0;
                            }

                            if (inter_bin_encoding >= 2) {
                                inter_off_on = 1;
                                inter_bin_encoding -= 2;
                            }
                            else {
                                inter_off_on = 0;
                            }

                            if (inter_bin_encoding == 1) inter_off_off = 1;
                            else inter_off_off = 0;

                            old_invalids_count = inter_on_on + inter_on_off + inter_off_on + inter_off_off;

                            if (lit1 < lit2) {
                                if (is_1_negative && is_2_negative) inter_on_on = 1;
                                else if (!is_1_negative && is_2_negative) inter_on_off = 1;
                                else if (is_1_negative && !is_2_negative) inter_off_on = 1;
                                else if (!is_1_negative && !is_2_negative) inter_off_off = 1;
                            }
                            else {
                                if (is_1_negative && is_2_negative) inter_on_on = 1;
                                else if (!is_1_negative && is_2_negative) inter_off_on = 1;
                                else if (is_1_negative && !is_2_negative) inter_on_off = 1;
                                else if (!is_1_negative && !is_2_negative) inter_off_off = 1;
                            }

                            new_invalids_count = inter_on_on + inter_on_off + inter_off_on + inter_off_off;

                            if (lit1 < lit2) invalids_uncovered[lit2 - 1] += new_invalids_count - old_invalids_count;
                            else invalids_uncovered[lit1 - 1] += new_invalids_count - old_invalids_count;

                            inter_bin_encoding = inter_off_off + 2 * inter_off_on + 4 * inter_on_off + 8 * inter_on_on;

                            invalid_feature_interactions[inter_index] = inter_bin_encoding;
                        }
                        else {
                            inter_on_on = 0;
                            inter_on_off = 0;
                            inter_off_on = 0;
                            inter_off_off = 0;

                            if (lit1 < lit2) {
                                if (is_1_negative && is_2_negative) inter_on_on = 1;
                                else if (!is_1_negative && is_2_negative) inter_on_off = 1;
                                else if (is_1_negative && !is_2_negative) inter_off_on = 1;
                                else if (!is_1_negative && !is_2_negative) inter_off_off = 1;
                            }
                            else {
                                if (is_1_negative && is_2_negative) inter_on_on = 1;
                                else if (!is_1_negative && is_2_negative) inter_off_on = 1;
                                else if (is_1_negative && !is_2_negative) inter_on_off = 1;
                                else if (!is_1_negative && !is_2_negative) inter_off_off = 1;
                            }

                            if (lit1 < lit2) invalids_uncovered[lit2 - 1]++;
                            else invalids_uncovered[lit1 - 1]++;

                            inter_bin_encoding = inter_off_off + 2 * inter_off_on + 4 * inter_on_off + 8 * inter_on_on;

                            invalid_feature_interactions[inter_index] = inter_bin_encoding;
                        }
                    }

                    if (clause_list_sizes[reduced_clause_count] > max_lits_per_clause) max_lits_per_clause = clause_list_sizes[reduced_clause_count];

                    if (clause_list_sizes[reduced_clause_count] > lits_in_clause_limit) {
                        cout << "Error: clause has more than " << lits_in_clause_limit << " (max.) literals! Exiting..." << endl;
                        exit(0);
                    }

                    total_lits_in_file += clause_list_sizes[reduced_clause_count];

                    reduced_clause_count++;

                    for (int i = 0; i < sampled_variants_size; i++) S[i].addClause_(lits);

                    if (reduced_clause_count % 5000 == 0) cout << "Loaded clause nr. " << reduced_clause_count << endl;
                }
            }
            else {
                if (lit_value > 0) cnf_command_params.push_back(lit_value);

                feature_count = cnf_command_params[0];
                init_clause_count = cnf_command_params[1];

                cout << "Feature / Init. clause count: " << feature_count << " / " << init_clause_count << endl;

                if (validity_checking) {
                    active_literals = (char*)malloc(feature_count * sizeof(char));
                    inactive_literals = (char*)malloc(feature_count * sizeof(char));

                    for (int i = 0; i < feature_count; i++) {
                        inactive_literals[i] = 0;
                        active_literals[i] = 0;

                        for (int j = 0; j < sampled_variants_size; j++) S[j].newVar();   
                    }
                }
                else {
                    for (int i = 0; i < feature_count; i++) {
                        for (int j = 0; j < sampled_variants_size; j++) S[j].newVar();
                    }
                }

                clause_list = (int*)malloc(init_clause_count * lits_in_clause_limit * sizeof(int));
                clause_list_sizes = (short*)malloc(init_clause_count * sizeof(short));

                feature_interaction_count = (feature_count * (feature_count - 1)) / 2;
                invalid_feature_interactions = (char*)malloc(feature_interaction_count * sizeof(char));
                invalids_uncovered = (int*)malloc((feature_count - 1) * sizeof(int));


                for (int i = 0; i < init_clause_count * lits_in_clause_limit; i++) clause_list[i] = 0;
                for (int i = 0; i < init_clause_count; i++) clause_list_sizes[i] = 0;
                for (int i = 0; i < feature_interaction_count; i++) invalid_feature_interactions[i] = 0;
                for (int i = 0; i < feature_count - 1; i++) invalids_uncovered[i] = 0;
            }
        }

        cnf_file.close();
    }
    else {
        cout << "ERROR! Cannot open file! Exiting..." << endl;
        exit(0);
    }

    cout << "\nReduced clause count: " << reduced_clause_count << endl;
    cout << "Max lits per clause: " << max_lits_per_clause << endl;
    cout << "Mean lits per clause: " << float(total_lits_in_file) / float(reduced_clause_count) << endl;
    
    unsigned long long invalids_total = 0;

    for (int i = 0; i < feature_count - 1; i++) {
        invalids_total += invalids_uncovered[i];
    }

    cout << "Found invalids: " << invalids_total << endl;

    clauses_with_lit = (int*)malloc(feature_count * sizeof(int));
    current_reference_iterators = (int*)malloc(feature_count * sizeof(int));
    reference_start_points = (unsigned long long*)malloc(feature_count * sizeof(unsigned long long));

    for (int i = 0; i < feature_count; i++) clauses_with_lit[i] = 0;

    for (int i = 0; i < reduced_clause_count * lits_in_clause_limit; i++) {
        if (clause_list[i] != 0) {
            lit_value = abs(clause_list[i]) - 1;
            clauses_with_lit[lit_value]++;
        }
    }

    int max_references_to_clauses = 0;
    unsigned long long cumulated_references_to_clauses = 0;

    for (int i = 0; i < feature_count; i++) {
        if (clauses_with_lit[i] > max_references_to_clauses) {
            max_references_to_clauses = clauses_with_lit[i];
        }

        reference_start_points[i] = cumulated_references_to_clauses;
        current_reference_iterators[i] = 0;

        cumulated_references_to_clauses += clauses_with_lit[i];
    }

    cout << "\nMax clauses: " << max_references_to_clauses << endl;
    cout << "Total lits in file: " << total_lits_in_file << endl;
    cout << "Mean clauses: " << float(total_lits_in_file) / float(feature_count) << endl;

    int clause_index, lit_index_i, lit_index_j;
    unsigned long long inter_index;

    feat_to_clause_reference = (int*)malloc(cumulated_references_to_clauses * sizeof(int));
    clause_to_feat_reference = (int*)malloc(cumulated_references_to_clauses * sizeof(int));
    are_feature_pairs_dependent = (char*)malloc(feature_interaction_count * sizeof(char));

    for (int i = 0; i < reduced_clause_count * lits_in_clause_limit; i++) {
        if (clause_list[i] != 0) {
            clause_index = (i - i % lits_in_clause_limit) / lits_in_clause_limit + 1;
            lit_value = abs(clause_list[i]) - 1;

            if (clause_list[i] > 0) feat_to_clause_reference[reference_start_points[lit_value] + current_reference_iterators[lit_value]] = clause_index;
            else feat_to_clause_reference[reference_start_points[lit_value] + current_reference_iterators[lit_value]] = -clause_index;
            current_reference_iterators[lit_value]++;
        }
    }

    for (int i = 0; i < feature_count; i++) {
        for (int j = 0; j < clauses_with_lit[i]; j++) {
            clause_to_feat_reference[reference_start_points[i] + j] = i;
        }
    }

    for (int i = 1; i < feature_count; i++) {
        for (int j = 0; j < i; j++) {
            inter_index = ((i - 1) * i) / 2 + j;
            are_feature_pairs_dependent[inter_index] = 0;
        }
    }

    for (int i = 0; i < reduced_clause_count; i++) {

        for (int j = 0; j < clause_list_sizes[i]; j++) {

            lit_index_i = clause_list[lits_in_clause_limit * i + j];
            lit_index_i = abs(lit_index_i) - 1;

            for (int k = 0; k < clause_list_sizes[i]; k++) {

                if (j != k) {
                    lit_index_j = clause_list[lits_in_clause_limit * i + k];
                    lit_index_j = abs(lit_index_j) - 1;

                    if (lit_index_i > lit_index_j) {
                        inter_index = ((lit_index_i - 1) * lit_index_i) / 2 + lit_index_j;
                        are_feature_pairs_dependent[inter_index] = 1;
                    }
                }
            }
        }
    }

    unsigned long long dependent_counter = 0;

    for (int i = 1; i < feature_count; i++) {
        for (int j = 0; j < i; j++) {
            inter_index = ((i - 1) * i) / 2 + j;
            if (are_feature_pairs_dependent[inter_index] == 1) dependent_counter++;
        }
    }

    cout << "Dependent interactions / total interactions: " << dependent_counter << " / " << feature_interaction_count << endl;

    auto elapsed0 = std::chrono::high_resolution_clock::now() - start0;

    long long microseconds0 = std::chrono::duration_cast<std::chrono::microseconds>(
        elapsed0).count();

    cout << "\nCNF file processing duration (us, ms, s): " << microseconds0 << ", " << microseconds0 / 1000 << ", " << microseconds0 / 1000000 << endl;

    cout << "\n=======================================\n" << endl;

    const int max_sample_size = 50000;

    if (sampled_variants_size > feature_count - 1) {
        cout << "ERROR! Sample size must be smaller than feature count. Exiting..." << endl;
        exit(0);
    }

    const int post_optimization_samples = 10;
    const bool disable_qip = false;
    const int optimization_epochs = 1000;
    const int max_retries_per_optimization = 25;
    const int min_retries_per_optimization = 7;
    int retries_per_optimization = max_retries_per_optimization;
    const bool dual_flip = false;
    const int step_size = 10;
    const bool modify_oldest = true;
    const bool limit_optimizations_to_sample_size = true;
    const int blockSize = 256;
    const int parallel_tasks = 100000;

    int numBlocksFeatures = int(round(static_cast<double>(feature_count + blockSize - 1) / static_cast<double>(blockSize)));
    int numBlocksClauses = int(round(static_cast<double>(reduced_clause_count + blockSize - 1) / static_cast<double>(blockSize)));

    int* current_sample_size, * current_sample_size_c;
    char* current_sample, * current_sample_c;
    int* skip_sample_index, * skip_sample_index_c;
    char* sampled_variants, * sampled_variants_c;
    unsigned long long* novel_fi_counts, * novel_fi_counts_c;
    unsigned long long* novel_fi_count, * novel_fi_count_c;
    char* feature_interactions, * feature_interactions_c;
    current_sample_size = (int*)malloc(sizeof(int));
    current_sample = (char*)malloc(feature_count * max_sample_size * sizeof(char));
    skip_sample_index = (int*)malloc(sizeof(int));
    sampled_variants = (char*)malloc(feature_count * sampled_variants_size * sizeof(char));
    novel_fi_counts = (unsigned long long*)malloc(sampled_variants_size * sizeof(unsigned long long));
    novel_fi_count = (unsigned long long*)malloc(sizeof(unsigned long long));
    feature_interactions = (char*)malloc(feature_interaction_count * sizeof(char));

    int* one_counts_c;

    int* max_index, * max_index_c;
    max_index = (int*)malloc(sizeof(int));

    int* max_index_m, * max_index_m_c;
    max_index_m = (int*)malloc(sizeof(int));

    int* max_gain_m;
    max_gain_m = (int*)malloc(sizeof(int));

    int* max_indices_j_m, * max_indices_j_m_c;
    max_indices_j_m = (int*)malloc(feature_count * sizeof(int));

    int* max_gains_m, * max_gains_m_c;
    max_gains_m = (int*)malloc(feature_count * sizeof(int));

    int* max_index_i_m, * max_index_i_m_c;
    max_index_i_m = (int*)malloc(sizeof(int));

    int* max_index_j_m, * max_index_j_m_c;
    max_index_j_m = (int*)malloc(sizeof(int));

    int* current_step_m, * current_step_m_c;
    current_step_m = (int*)malloc(sizeof(int));

    int* M_values, * M_values_c;
    M_values = (int*)malloc(feature_count * sizeof(int));

    char* optimized_variant, * optimized_variant_temp, * optimized_variant_c;
    optimized_variant = (char*)malloc(feature_count * sizeof(char));
    optimized_variant_temp = (char*)malloc(feature_count * sizeof(char));

    int* uncovered, * uncovered_c;
    uncovered = (int*)malloc((feature_count - 1) * sizeof(int));

    int* variant_ages;
    variant_ages = (int*)malloc(max_sample_size * sizeof(int));

    int* is_flip_valid, * is_flip_valid_c;
    is_flip_valid = (int*)malloc(feature_count * sizeof(int));

    int* shuffled_features;
    shuffled_features = (int*)malloc((feature_count - 1) * sizeof(int));

    unsigned long long* invalids_total_list;
    invalids_total_list = (unsigned long long*)malloc((sampled_variants_size) * sizeof(unsigned long long));

    hipMalloc(&clause_list_c, init_clause_count * lits_in_clause_limit * sizeof(int));
    hipMalloc(&clause_list_sizes_c, init_clause_count * sizeof(short));
    hipMalloc(&current_clause_values_c, reduced_clause_count * sizeof(short));
    hipMalloc(&min_clause_values_c, reduced_clause_count * sizeof(short));

    hipMalloc(&clauses_with_lit_c, feature_count * sizeof(int));
    hipMalloc(&reference_start_points_c, feature_count * sizeof(unsigned long long));
    hipMalloc(&feat_to_clause_reference_c, cumulated_references_to_clauses * sizeof(int));
    hipMalloc(&clause_to_feat_reference_c, cumulated_references_to_clauses * sizeof(int));

    hipMalloc(&is_flip_valid_c, feature_count * sizeof(int));
    hipMalloc(&dual_invalids_c, feature_interaction_count * sizeof(int));
    hipMalloc(&dual_valids_c, feature_interaction_count * sizeof(int));
    hipMalloc(&valid_thresholds_c, feature_count * sizeof(int));
    hipMalloc(&are_feature_pairs_dependent_c, feature_interaction_count * sizeof(char));

    hipMalloc(&current_sample_size_c, sizeof(int));
    hipMalloc(&current_sample_c, feature_count * max_sample_size * sizeof(char));
    hipMalloc(&skip_sample_index_c, sizeof(int));
    hipMalloc(&sampled_variants_c, feature_count * sampled_variants_size * sizeof(char));
    hipMalloc(&novel_fi_counts_c, sampled_variants_size * sizeof(unsigned long long));
    hipMalloc(&novel_fi_count_c, sizeof(unsigned long long));
    hipMalloc(&feature_interactions_c, feature_interaction_count * sizeof(char));
    hipMalloc(&one_counts_c, feature_count * sizeof(int));
    hipMalloc(&max_index_c, sizeof(int));
    hipMalloc(&max_index_m_c, sizeof(int));
    hipMalloc(&current_step_m_c, sizeof(int));

    hipMalloc(&max_indices_j_m_c, feature_count * sizeof(int));
    hipMalloc(&max_gains_m_c, feature_count * sizeof(int));

    hipMalloc(&max_index_i_m_c, sizeof(int));
    hipMalloc(&max_index_j_m_c, sizeof(int));

    hipMalloc(&M_values_c, feature_count * sizeof(int));
    hipMalloc(&optimized_variant_c, feature_count * sizeof(char));
    hipMalloc(&uncovered_c, (feature_count - 1) * sizeof(int));

    hiprandState* rand_state;
    hipMalloc(&rand_state, feature_count * sizeof(hiprandState));

    char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
    char inter_bin_encoding;
    char higher_value, lower_value;

    double rand_num;
    current_sample_size[0] = 1;

    for (int i = 0; i < feature_count * max_sample_size; i++) {
        if (i < feature_count * current_sample_size[0]) {
            rand_num = unif(rng);
            if (rand_num > 0.5) current_sample[i] = 1;
            else current_sample[i] = 0;
        }
        else current_sample[i] = 0;
    }

    S[0].resetSolver(feature_count);
    S[0].resetSimpSolver(feature_count);

    S[0].setTargetValue(0, feature_count, current_sample);

    S[0].eliminate(true);

    if (!S[0].okay()) {
        printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
        exit(0);
    }

    vec<Lit> dummy0;
    lbool ret0 = S[0].solveLimited(dummy0);

    if (ret0 == l_True) {
        for (int i = 0; i < feature_count; i++) {
            if (S[0].model[i] != l_Undef) {
                if (S[0].model[i] == l_True) current_sample[i] = 1;
                else current_sample[i] = 0;
            }
            else {
                printf("UNDEFINED VARIABLES, exiting...\n");
                exit(0);
            }
        }
    }
    else {
        printf("UNSATISFIABLE, exiting...\n");
        exit(0);
    }

    int age_counter = 1;
    variant_ages[0] = age_counter;
    age_counter++;

    skip_sample_index[0] = -1;

    std::thread myThreads[sampled_variants_size];

    for (int i = 0; i < feature_count - 1; i++) {
        shuffled_features[i] = i;
    }

    auto start1 = std::chrono::high_resolution_clock::now();

    hipMemcpy(clause_list_c, clause_list, init_clause_count * lits_in_clause_limit * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(clause_list_sizes_c, clause_list_sizes, init_clause_count * sizeof(short), hipMemcpyHostToDevice);
    hipMemcpy(clauses_with_lit_c, clauses_with_lit, feature_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(reference_start_points_c, reference_start_points, feature_count * sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(feat_to_clause_reference_c, feat_to_clause_reference, cumulated_references_to_clauses * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(clause_to_feat_reference_c, clause_to_feat_reference, cumulated_references_to_clauses * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(are_feature_pairs_dependent_c, are_feature_pairs_dependent, feature_interaction_count * sizeof(char), hipMemcpyHostToDevice);

    hipMemcpy(current_sample_size_c, current_sample_size, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(current_sample_c, current_sample, feature_count * max_sample_size * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(skip_sample_index_c, skip_sample_index, sizeof(int), hipMemcpyHostToDevice);

    reset_uncovered << < numBlocksFeatures, blockSize >> > (feature_count, uncovered_c);
    calculate_interactions << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, current_sample_size_c, current_sample_c, skip_sample_index_c, feature_interactions_c, uncovered_c); // Threads: max. 1024

    vector<unsigned long long> uncovered_fi;
    vector<unsigned long long> invalid_fi;
    vector<int> variant_id;

    int variant_counter = 1;

    for (int n = 2; n < max_sample_size - 1; n++) {
        const int a = int(round(unif2(rng)));
        const int b = int(round(unif2(rng)));

        init_rand << < numBlocksFeatures, blockSize >> > (rand_state, a, b, feature_count);

        hipMemcpy(uncovered, uncovered_c, (feature_count - 1) * sizeof(int), hipMemcpyDeviceToHost);

        unsigned long long uncovered_total = 0;

        for (int i = 0; i < feature_count - 1; i++)  uncovered_total += uncovered[i];

        cout << ">>>>>> Uncovered interactions: " << uncovered_total << "  Invalid interactions: " << invalids_total << " <<<<<< " << endl;

        uncovered_fi.push_back(uncovered_total);
        invalid_fi.push_back(invalids_total);
        variant_id.push_back(variant_counter);
        variant_counter++;

        if (uncovered_total == invalids_total) {
            cout << "===============================" << endl;
            cout << "End reached...\n" << endl;
            break;
        }

        calculate_feature_probability << < numBlocksFeatures, blockSize >> > (feature_count, current_sample_size_c, current_sample_c, one_counts_c);
        init_sample << < numBlocksFeatures, blockSize >> > (rand_state, feature_count, current_sample_size_c, sampled_variants_size, sampled_variants_c, one_counts_c);

        hipMemcpy(sampled_variants, sampled_variants_c, feature_count * sampled_variants_size * sizeof(char), hipMemcpyDeviceToHost);
        hipMemcpy(feature_interactions, feature_interactions_c, feature_interaction_count * sizeof(char), hipMemcpyDeviceToHost);
        hipMemcpy(feature_interactions, feature_interactions_c, feature_interaction_count * sizeof(char), hipMemcpyDeviceToHost);

        random_shuffle(&shuffled_features[0], &shuffled_features[feature_count - 1]);

        cout << "Solver:" << endl;

        int start_point_x = 0, start_point_j = 0;

        bool found_valid, skip, end_reached = false;
        unsigned long long inter_index;
        char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
        char invalid_inter_off_off, invalid_inter_off_on, invalid_inter_on_off, invalid_inter_on_on;
        char inter_bin_encoding, invalid_inter_bin_encoding;

        for (int m = 0; m < sampled_variants_size; m++) {

            skip = false;

            found_valid = false;

            if (!end_reached) {
                if (start_point_x != 0 || start_point_j != 0) skip = true;

                for (int x = start_point_x; x < feature_count - 1; x++) {

                    int i = shuffled_features[x];

                    if (invalids_uncovered[i] < uncovered[i]) {
                        for (int j = start_point_j; j < i + 1; j++) {

                            if (!skip) {
                                inter_index = (i * (i + 1)) / 2 + j;

                                inter_bin_encoding = feature_interactions[inter_index];
                                invalid_inter_bin_encoding = invalid_feature_interactions[inter_index];

                                if (inter_bin_encoding != 0 && inter_bin_encoding != invalid_inter_bin_encoding) {

                                    if (inter_bin_encoding >= 8) {
                                        inter_on_on = 1;
                                        inter_bin_encoding -= 8;
                                    }
                                    else {
                                        inter_on_on = 0;
                                    }

                                    if (inter_bin_encoding >= 4) {
                                        inter_on_off = 1;
                                        inter_bin_encoding -= 4;
                                    }
                                    else {
                                        inter_on_off = 0;
                                    }

                                    if (inter_bin_encoding >= 2) {
                                        inter_off_on = 1;
                                        inter_bin_encoding -= 2;
                                    }
                                    else {
                                        inter_off_on = 0;
                                    }

                                    if (inter_bin_encoding == 1) inter_off_off = 1;
                                    else inter_off_off = 0;

                                    if (invalid_inter_bin_encoding >= 8) {
                                        invalid_inter_on_on = 1;
                                        invalid_inter_bin_encoding -= 8;
                                    }
                                    else {
                                        invalid_inter_on_on = 0;
                                    }

                                    if (invalid_inter_bin_encoding >= 4) {
                                        invalid_inter_on_off = 1;
                                        invalid_inter_bin_encoding -= 4;
                                    }
                                    else {
                                        invalid_inter_on_off = 0;
                                    }

                                    if (invalid_inter_bin_encoding >= 2) {
                                        invalid_inter_off_on = 1;
                                        invalid_inter_bin_encoding -= 2;
                                    }
                                    else {
                                        invalid_inter_off_on = 0;
                                    }

                                    if (invalid_inter_bin_encoding == 1) invalid_inter_off_off = 1;
                                    else invalid_inter_off_off = 0;

                                    if (invalid_inter_on_on == 0 && inter_on_on == 1) {
                                        S[0].resetSolver(feature_count);
                                        S[0].resetSimpSolver(feature_count);

                                        S[0].setTargetValue(feature_count * m, feature_count, sampled_variants);

                                        S[0].eliminate(true);

                                        if (!S[0].okay()) {
                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                            exit(0);
                                        }

                                        vec<Lit> dummy1;

                                        dummy1.push(mkLit(i + 1));
                                        dummy1.push(mkLit(j));

                                        lbool ret1 = S[0].solveLimited(dummy1);

                                        if (ret1 == l_True) {
                                            for (int k = 0; k < feature_count; k++) {
                                                if (S[0].model[k] != l_Undef) {
                                                    if (S[0].model[k] == l_True) sampled_variants[m * feature_count + k] = 1;
                                                    else sampled_variants[m * feature_count + k] = 0;
                                                }
                                                else {
                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                    exit(0);
                                                }
                                            }

                                            start_point_x = x;
                                            start_point_j = j;

                                            found_valid = true;
                                            break;
                                        }
                                        else {
                                            invalid_inter_on_on = 1;

                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                            invalids_uncovered[i]++;

                                            invalids_total++;

                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                        }
                                    }

                                    if (invalid_inter_on_off == 0 && inter_on_off == 1) {
                                        S[0].resetSolver(feature_count);
                                        S[0].resetSimpSolver(feature_count);

                                        S[0].setTargetValue(feature_count * m, feature_count, sampled_variants);

                                        S[0].eliminate(true);

                                        if (!S[0].okay()) {
                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                            exit(0);
                                        }

                                        vec<Lit> dummy1;

                                        dummy1.push(mkLit(i + 1));
                                        dummy1.push(~mkLit(j));

                                        lbool ret1 = S[0].solveLimited(dummy1);
                                        if (ret1 == l_True) {
                                            for (int k = 0; k < feature_count; k++) {
                                                if (S[0].model[k] != l_Undef) {
                                                    if (S[0].model[k] == l_True) sampled_variants[m * feature_count + k] = 1;
                                                    else sampled_variants[m * feature_count + k] = 0;
                                                }
                                                else {
                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                    exit(0);
                                                }
                                            }

                                            start_point_x = x;
                                            start_point_j = j;

                                            found_valid = true;
                                            break;
                                        }
                                        else {
                                            invalid_inter_on_off = 1;

                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                            invalids_uncovered[i]++;

                                            invalids_total++;

                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                        }
                                    }

                                    if (invalid_inter_off_on == 0 && inter_off_on == 1) {
                                        S[0].resetSolver(feature_count);
                                        S[0].resetSimpSolver(feature_count);

                                        S[0].setTargetValue(feature_count * m, feature_count, sampled_variants);

                                        S[0].eliminate(true);

                                        if (!S[0].okay()) {
                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                            exit(0);
                                        }

                                        vec<Lit> dummy1;

                                        dummy1.push(~mkLit(i + 1));
                                        dummy1.push(mkLit(j));

                                        lbool ret1 = S[0].solveLimited(dummy1);
                                        if (ret1 == l_True) {
                                            for (int k = 0; k < feature_count; k++) {
                                                if (S[0].model[k] != l_Undef) {
                                                    if (S[0].model[k] == l_True) sampled_variants[m * feature_count + k] = 1;
                                                    else sampled_variants[m * feature_count + k] = 0;
                                                }
                                                else {
                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                    exit(0);
                                                }
                                            }

                                            start_point_x = x;
                                            start_point_j = j;

                                            found_valid = true;
                                            break;
                                        }
                                        else {
                                            invalid_inter_off_on = 1;

                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                            invalids_uncovered[i]++;

                                            invalids_total++;

                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                        }
                                    }

                                    if (invalid_inter_off_off == 0 && inter_off_off == 1) {
                                        S[0].resetSolver(feature_count);
                                        S[0].resetSimpSolver(feature_count);

                                        S[0].setTargetValue(feature_count * m, feature_count, sampled_variants);

                                        S[0].eliminate(true);

                                        if (!S[0].okay()) {
                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                            exit(0);
                                        }

                                        vec<Lit> dummy1;

                                        dummy1.push(~mkLit(i + 1));
                                        dummy1.push(~mkLit(j));

                                        lbool ret1 = S[0].solveLimited(dummy1);
                                        if (ret1 == l_True) {
                                            for (int k = 0; k < feature_count; k++) {
                                                if (S[0].model[k] != l_Undef) {
                                                    if (S[0].model[k] == l_True) sampled_variants[m * feature_count + k] = 1;
                                                    else sampled_variants[m * feature_count + k] = 0;
                                                }
                                                else {
                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                    exit(0);
                                                }
                                            }

                                            start_point_x = x;
                                            start_point_j = j;

                                            found_valid = true;
                                            break;
                                        }
                                        else {
                                            invalid_inter_off_off = 1;

                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                            invalids_uncovered[i]++;

                                            invalids_total++;

                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                        }
                                    }

                                }
                            }
                            else skip = false;
                        }
                    }
                    if (found_valid) break;
                }
            }

            if (!found_valid) {
                end_reached = true;

                S[0].resetSolver(feature_count);
                S[0].resetSimpSolver(feature_count);

                S[0].setTargetValue(feature_count * m, feature_count, sampled_variants);

                S[0].eliminate(true);

                if (!S[0].okay()) {
                    printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                    exit(0);
                }

                vec<Lit> dummy1;
                lbool ret1 = S[0].solveLimited(dummy1);

                if (ret1 == l_True) {

                    for (int k = 0; k < feature_count; k++) {
                        if (S[0].model[k] != l_Undef) {
                            if (S[0].model[k] == l_True) sampled_variants[m * feature_count + k] = 1;
                            else sampled_variants[m * feature_count + k] = 0;
                        }
                        else {
                            printf("UNDEFINED VARIABLES, exiting...\n");
                            exit(0);
                        }
                    }
                }
                else {
                    printf("UNSAT, exiting...\n");
                    exit(0);
                }
            }
        }

        if (uncovered_total == invalids_total) {
            cout << "===============================" << endl;
            cout << "End reached...\n" << endl;
            break;
        }

        hipMemcpy(sampled_variants_c, sampled_variants, feature_count * sampled_variants_size * sizeof(char), hipMemcpyHostToDevice);

        reset_novel_fi_counts << < 1, 1 >> > (sampled_variants_size, novel_fi_counts_c);
        calculate_sample_gain_all << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, sampled_variants_size, sampled_variants_c, novel_fi_counts_c, feature_interactions_c);

        find_max_sample << < 1, 1 >> > (sampled_variants_size, novel_fi_counts_c, max_index_c);
        write_max_sample_to_optimized << < numBlocksFeatures, blockSize >> > (feature_count, sampled_variants_c, optimized_variant_c, max_index_c);

        reset_M_values << < numBlocksFeatures, blockSize >> > (feature_count, M_values_c);
        calculate_init_M_values << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interactions_c, optimized_variant_c, M_values_c);

        calculate_clause_values << < numBlocksClauses, blockSize >> > (reduced_clause_count, current_clause_values_c, min_clause_values_c, clause_list_c, lits_in_clause_limit, clause_list_sizes_c, optimized_variant_c);

        int it_count = 0;
        int zero_counter = 0;
        unsigned long long total_gain = 0;

        while (it_count < optimization_epochs && !disable_qip) {

            if (!dual_flip) {

                reset_flip_counts << < numBlocksFeatures, blockSize >> > (feature_count, is_flip_valid_c);

                calculate_flip_validity_optimized << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, total_lits_in_file, current_clause_values_c, min_clause_values_c, feat_to_clause_reference_c, clause_to_feat_reference_c, optimized_variant_c, is_flip_valid_c);

                hipMemcpy(M_values, M_values_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(optimized_variant, optimized_variant_c, feature_count * sizeof(char), hipMemcpyDeviceToHost);
                hipMemcpy(is_flip_valid, is_flip_valid_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);

                max_index_m[0] = -1;
                max_gain_m[0] = 0;

                int gain_value;
                char value;

                for (int i = 0; i < feature_count; i++) {
                    if (is_flip_valid[i] == 0) {
                        value = optimized_variant[i];
                        gain_value = M_values[i] * (1 - 2 * value);

                        if (gain_value > max_gain_m[0]) {
                            max_gain_m[0] = gain_value;
                            max_index_m[0] = i;
                        }
                    }
                }

                if (max_gain_m[0] > 0) {
                    hipMemcpy(max_index_m_c, max_index_m, sizeof(int), hipMemcpyHostToDevice);

                    adapt_M_values << < numBlocksFeatures, blockSize >> > (feature_count, feature_interactions_c, optimized_variant_c, M_values_c, max_index_m_c);

                    int numBlocksAffectedClauses = int(round(static_cast<double>(clauses_with_lit[max_index_m[0]]) + blockSize - 1) / static_cast<double>(blockSize));

                    adapt_clause_values << < numBlocksAffectedClauses, blockSize >> > (current_clause_values_c, clauses_with_lit_c, feat_to_clause_reference_c, reference_start_points_c, optimized_variant_c, max_index_m_c);

                    optimize_variant << < 1, 1 >> > (optimized_variant_c, max_index_m_c);
                    total_gain += max_gain_m[0];
                }
                else break;

            }
            else {
                current_step_m[0] = it_count % step_size;

                hipMemcpy(current_step_m_c, current_step_m, sizeof(int), hipMemcpyHostToDevice);

                reset_dual_flip_counts << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, is_flip_valid_c, dual_invalids_c, dual_valids_c, valid_thresholds_c);

                calculate_dual_flip_validity_optimized << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, total_lits_in_file, current_clause_values_c, min_clause_values_c, feat_to_clause_reference_c, clause_to_feat_reference_c, clause_list_c, clause_list_sizes_c, lits_in_clause_limit, optimized_variant_c, is_flip_valid_c, dual_invalids_c, dual_valids_c, valid_thresholds_c);

                calculate_max_gain_dual_with_validity_check_2 << < numBlocksFeatures, blockSize >> > (feature_count, feature_interactions_c, optimized_variant_c, M_values_c, max_indices_j_m_c, max_gains_m_c, step_size, current_step_m_c, is_flip_valid_c, dual_invalids_c, dual_valids_c, valid_thresholds_c, are_feature_pairs_dependent_c);

                hipMemcpy(max_gains_m, max_gains_m_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(max_indices_j_m, max_indices_j_m_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);
                
                int gain_value = 0;

                for (int i = 0; i < feature_count; i++) {
                    if (max_gains_m[i] > gain_value) {
                        gain_value = max_gains_m[i];
                        max_index_i_m[0] = i;
                        max_index_j_m[0] = max_indices_j_m[i];
                    }
                }

                if (gain_value > 0) {
                    hipMemcpy(max_index_i_m_c, max_index_i_m, sizeof(int), hipMemcpyHostToDevice);
                    hipMemcpy(max_index_j_m_c, max_index_j_m, sizeof(int), hipMemcpyHostToDevice);

                    adapt_M_values_dual << < numBlocksFeatures, blockSize >> > (feature_count, feature_interactions_c, optimized_variant_c, M_values_c, max_index_i_m_c, max_index_j_m_c);

                    int numBlocksAffectedClauses = int(round(static_cast<double>(clauses_with_lit[max_index_i_m[0]]) + blockSize - 1) / static_cast<double>(blockSize));

                    adapt_clause_values << < numBlocksAffectedClauses, blockSize >> > (current_clause_values_c, clauses_with_lit_c, feat_to_clause_reference_c, reference_start_points_c, optimized_variant_c, max_index_i_m_c);

                    if (max_index_j_m[0] != -1) {
                        numBlocksAffectedClauses = int(round(static_cast<double>(clauses_with_lit[max_index_j_m[0]]) + blockSize - 1) / static_cast<double>(blockSize));
                        adapt_clause_values << < numBlocksAffectedClauses, blockSize >> > (current_clause_values_c, clauses_with_lit_c, feat_to_clause_reference_c, reference_start_points_c, optimized_variant_c, max_index_j_m_c);
                    }

                    optimize_variant_dual << < 1, 1 >> > (optimized_variant_c, max_index_i_m_c, max_index_j_m_c);

                    total_gain += gain_value;

                    zero_counter = 0;
                }
                else {
                    zero_counter++;

                    if (zero_counter == step_size) break;
                }
            }

            it_count++;
        }

        /*
        hipMemcpy(optimized_variant, optimized_variant_c, feature_count * sizeof(char), hipMemcpyDeviceToHost);

        ifstream file1(cnf_file_name);

        bool found;
        int sat_counter = 0;
        int unsat_counter = 0;
        int number;
        bool negative, comment, command;

        if (file1.is_open()) {
            while (getline(file1, file_line)) {
                number = 0;
                negative = false;
                found = false;

                comment = false;
                command = false;

                char_counter = 0;

                for (char& c : file_line) {
                    int index = int(c);

                    if (char_counter == 0 && index == 99) {
                        comment = true;
                        break;
                    }

                    if (char_counter == 0 && index == 112) {
                        command = true;
                        break;
                    }

                    if (index >= 48 && index <= 57) {
                        if (number == 0) {
                            number = index - 48;
                        }
                        else {
                            number *= 10;
                            number += index - 48;
                        }
                    }
                    else if (index == 45) {
                        negative = true;
                    }
                    else {
                        if (number > 0) {
                            if (negative) {
                                if (optimized_variant[number - 1] == 0) found = true;
                            }
                            else {
                                if (optimized_variant[number - 1] == 1) found = true;
                            }
                        }

                        number = 0;
                        negative = false;
                    }

                    char_counter++;
                }

                if (comment || command) continue;

                if (number > 0) {
                    if (negative) {
                        if (optimized_variant[number - 1] == 0) found = true;
                    }
                    else {
                        if (optimized_variant[number - 1] == 1) found = true;
                    }
                }

                if (!found) unsat_counter++;
                else sat_counter++;
            }

            file1.close();
        }

        std::cout << "\nCLAUSES SAT / UNSAT: " << sat_counter << " / " << unsat_counter << std::endl;
        */

        hipMemcpy(novel_fi_counts, novel_fi_counts_c, sampled_variants_size * sizeof(unsigned long long), hipMemcpyDeviceToHost);

        hipMemcpy(max_index, max_index_c, sizeof(int), hipMemcpyDeviceToHost);

        write_optimized_to_current_sample << < numBlocksFeatures, blockSize >> > (feature_count, optimized_variant_c, current_sample_size_c, current_sample_c);

        if (post_optimization_samples == 0) append_interactions << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, optimized_variant_c, feature_interactions_c, uncovered_c);

        increase_current_sample_size << < 1, 1 >> > (current_sample_size_c);

        if (modify_oldest) {
            variant_ages[n - 1] = age_counter;
            age_counter++;
        }

        cout << ">>>>>> Variant " << n << " <<<<<<" << endl;

        cout << "New covered interactions before: " << novel_fi_counts[max_index[0]] << " - Optimized: " << novel_fi_counts[max_index[0]] + total_gain << " Factor: " << float(novel_fi_counts[max_index[0]] + total_gain) / float(novel_fi_counts[max_index[0]]) << endl;

        cout << "-------------------------------" << endl;

        uncovered_total -= novel_fi_counts[max_index[0]] + total_gain;

        if (post_optimization_samples > 0) {

            std::uniform_real_distribution<double> unif3(0, n - 1);

            int limit;
            int not_zero_counter = 0;

            if (limit_optimizations_to_sample_size) limit = std::min(post_optimization_samples, n);
            else limit = post_optimization_samples;

            for (int m = 0; m < limit; m++) {

                if (modify_oldest) {
                    int min_age = age_counter;

                    for (int i = 0; i < n; i++) {
                        if (variant_ages[i] < min_age) {
                            skip_sample_index[0] = i;
                            min_age = variant_ages[i];
                        }
                    }
                }
                else skip_sample_index[0] = int(round(unif3(rng)));

                hipMemcpy(skip_sample_index_c, skip_sample_index, sizeof(int), hipMemcpyHostToDevice);

                reset_uncovered << < numBlocksFeatures, blockSize >> > (feature_count, uncovered_c);
                calculate_interactions << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, current_sample_size_c, current_sample_c, skip_sample_index_c, feature_interactions_c, uncovered_c); // Threads: max. 1024

                hipMemcpy(uncovered, uncovered_c, (feature_count - 1) * sizeof(int), hipMemcpyDeviceToHost);

                unsigned long long uncovered_without = 0;

                for (int i = 0; i < feature_count - 1; i++) {
                    uncovered_without += uncovered[i];
                }

                write_selected_sample_to_optimized << < numBlocksFeatures, blockSize >> > (feature_count, current_sample_c, optimized_variant_c, skip_sample_index_c);

                reset_M_values << < numBlocksFeatures, blockSize >> > (feature_count, M_values_c);
                calculate_init_M_values << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interactions_c, optimized_variant_c, M_values_c);

                calculate_clause_values << < numBlocksClauses, blockSize >> > (reduced_clause_count, current_clause_values_c, min_clause_values_c, clause_list_c, lits_in_clause_limit, clause_list_sizes_c, optimized_variant_c);

                it_count = 0;

                total_gain = 0;

                zero_counter = 0;

                int run_counter = 0;

                int start_point_x = 0, start_point_j = 0;

                random_shuffle(&shuffled_features[0], &shuffled_features[feature_count - 1]);

                bool use_alternative = false;

                while (it_count < optimization_epochs) {

                    if (!dual_flip) {
                        
                        if (!disable_qip) {
                            reset_flip_counts << < numBlocksFeatures, blockSize >> > (feature_count, is_flip_valid_c);

                            calculate_flip_validity_optimized << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, total_lits_in_file, current_clause_values_c, min_clause_values_c, feat_to_clause_reference_c, clause_to_feat_reference_c, optimized_variant_c, is_flip_valid_c);

                            hipMemcpy(M_values, M_values_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);
                            hipMemcpy(optimized_variant, optimized_variant_c, feature_count * sizeof(char), hipMemcpyDeviceToHost);
                            hipMemcpy(is_flip_valid, is_flip_valid_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);

                            max_index_m[0] = -1;
                            max_gain_m[0] = 0;

                            int gain_value;
                            char value;

                            for (int i = 0; i < feature_count; i++) {
                                if (is_flip_valid[i] == 0) {
                                    value = optimized_variant[i];

                                    gain_value = M_values[i] * (1 - 2 * value);

                                    if (gain_value > max_gain_m[0]) {
                                        max_gain_m[0] = gain_value;
                                        max_index_m[0] = i;
                                    }
                                }
                            }
                        } else max_gain_m[0] = 0;

                        if (max_gain_m[0] > 0) {
                            hipMemcpy(max_index_m_c, max_index_m, sizeof(int), hipMemcpyHostToDevice);

                            adapt_M_values << < numBlocksFeatures, blockSize >> > (feature_count, feature_interactions_c, optimized_variant_c, M_values_c, max_index_m_c);

                            int numBlocksAffectedClauses = int(round(static_cast<double>(clauses_with_lit[max_index_m[0]]) + blockSize - 1) / static_cast<double>(blockSize));

                            adapt_clause_values << < numBlocksAffectedClauses, blockSize >> > (current_clause_values_c, clauses_with_lit_c, feat_to_clause_reference_c, reference_start_points_c, optimized_variant_c, max_index_m_c);

                            optimize_variant << < 1, 1 >> > (optimized_variant_c, max_index_m_c);
                            
                            if (run_counter == 0) total_gain += max_gain_m[0];
                        }
                        else {
                            if (total_gain > 0) break;

                            if (run_counter == 0) {
                                hipMemcpy(optimized_variant, optimized_variant_c, feature_count * sizeof(char), hipMemcpyDeviceToHost);
                                hipMemcpy(feature_interactions, feature_interactions_c, feature_interaction_count * sizeof(char), hipMemcpyDeviceToHost);

                                for (int i = 0; i < feature_count; i++) optimized_variant_temp[i] = optimized_variant[i];
                            }

                            if (run_counter < retries_per_optimization) {

                                if (run_counter >= 1) {
                                    reset_novel_fi_count << < 1, 1 >> > (novel_fi_count_c);
                                    calculate_sample_gain_optimized << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, optimized_variant_c, novel_fi_count_c, feature_interactions_c);
                                    hipMemcpy(novel_fi_count, novel_fi_count_c, sizeof(unsigned long long), hipMemcpyDeviceToHost);

                                    if (novel_fi_count[0] > uncovered_without - uncovered_total + total_gain) {
                                        cout << "Found something better!" << endl;
                                        use_alternative = true;
                                        break;
                                    }
                                }

                                run_counter++;

                                bool skip = false;

                                if (start_point_x != 0 || start_point_j != 0) skip = true;

                                bool found_valid = false;
                                unsigned long long inter_index;
                                char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
                                char invalid_inter_off_off, invalid_inter_off_on, invalid_inter_on_off, invalid_inter_on_on;
                                char inter_bin_encoding, invalid_inter_bin_encoding;

                                for (int x = start_point_x; x < feature_count - 1; x++) {

                                    int i = shuffled_features[x];

                                    if (invalids_uncovered[i] < uncovered[i]) {
                                        for (int j = start_point_j; j < i + 1; j++) {

                                            if (!skip) {
                                                inter_index = (i * (i + 1)) / 2 + j;

                                                inter_bin_encoding = feature_interactions[inter_index];
                                                invalid_inter_bin_encoding = invalid_feature_interactions[inter_index];

                                                if (inter_bin_encoding != 0 && inter_bin_encoding != invalid_inter_bin_encoding) {

                                                    if (inter_bin_encoding >= 8) {
                                                        inter_on_on = 1;
                                                        inter_bin_encoding -= 8;
                                                    }
                                                    else {
                                                        inter_on_on = 0;
                                                    }

                                                    if (inter_bin_encoding >= 4) {
                                                        inter_on_off = 1;
                                                        inter_bin_encoding -= 4;
                                                    }
                                                    else {
                                                        inter_on_off = 0;
                                                    }

                                                    if (inter_bin_encoding >= 2) {
                                                        inter_off_on = 1;
                                                        inter_bin_encoding -= 2;
                                                    }
                                                    else {
                                                        inter_off_on = 0;
                                                    }

                                                    if (inter_bin_encoding == 1) inter_off_off = 1;
                                                    else inter_off_off = 0;

                                                    if (invalid_inter_bin_encoding >= 8) {
                                                        invalid_inter_on_on = 1;
                                                        invalid_inter_bin_encoding -= 8;
                                                    }
                                                    else {
                                                        invalid_inter_on_on = 0;
                                                    }

                                                    if (invalid_inter_bin_encoding >= 4) {
                                                        invalid_inter_on_off = 1;
                                                        invalid_inter_bin_encoding -= 4;
                                                    }
                                                    else {
                                                        invalid_inter_on_off = 0;
                                                    }

                                                    if (invalid_inter_bin_encoding >= 2) {
                                                        invalid_inter_off_on = 1;
                                                        invalid_inter_bin_encoding -= 2;
                                                    }
                                                    else {
                                                        invalid_inter_off_on = 0;
                                                    }

                                                    if (invalid_inter_bin_encoding == 1) invalid_inter_off_off = 1;
                                                    else invalid_inter_off_off = 0;

                                                    if (invalid_inter_on_on == 0 && inter_on_on == 1 && !(optimized_variant_temp[i + 1] == 1 && optimized_variant_temp[j] == 1)) {
                                                        S[0].resetSolver(feature_count);
                                                        S[0].resetSimpSolver(feature_count);

                                                        S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                        S[0].eliminate(true);

                                                        if (!S[0].okay()) {
                                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                            exit(0);
                                                        }

                                                        vec<Lit> dummy1;

                                                        dummy1.push(mkLit(i + 1));
                                                        dummy1.push(mkLit(j));

                                                        lbool ret1 = S[0].solveLimited(dummy1);

                                                        if (ret1 == l_True) {
                                                            for (int k = 0; k < feature_count; k++) {
                                                                if (S[0].model[k] != l_Undef) {
                                                                    if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                    else optimized_variant[k] = 0;
                                                                }
                                                                else {
                                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                                    exit(0);
                                                                }
                                                            }

                                                            start_point_x = x;
                                                            start_point_j = j;

                                                            found_valid = true;
                                                            break;
                                                        }
                                                        else {
                                                            invalid_inter_on_on = 1;

                                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                            invalids_uncovered[i]++;

                                                            invalids_total++;

                                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                                        }
                                                    }

                                                    if (invalid_inter_on_off == 0 && inter_on_off == 1 && !(optimized_variant_temp[i + 1] == 1 && optimized_variant_temp[j] == 0)) {
                                                        S[0].resetSolver(feature_count);
                                                        S[0].resetSimpSolver(feature_count);

                                                        S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                        S[0].eliminate(true);

                                                        if (!S[0].okay()) {
                                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                            exit(0);
                                                        }

                                                        vec<Lit> dummy1;

                                                        dummy1.push(mkLit(i + 1));
                                                        dummy1.push(~mkLit(j));

                                                        lbool ret1 = S[0].solveLimited(dummy1);
                                                        if (ret1 == l_True) {
                                                            for (int k = 0; k < feature_count; k++) {
                                                                if (S[0].model[k] != l_Undef) {
                                                                    if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                    else optimized_variant[k] = 0;
                                                                }
                                                                else {
                                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                                    exit(0);
                                                                }
                                                            }

                                                            start_point_x = x;
                                                            start_point_j = j;

                                                            found_valid = true;
                                                            break;
                                                        }
                                                        else {
                                                            invalid_inter_on_off = 1;

                                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                            invalids_uncovered[i]++;

                                                            invalids_total++;

                                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                                        }
                                                    }

                                                    if (invalid_inter_off_on == 0 && inter_off_on == 1 && !(optimized_variant_temp[i + 1] == 0 && optimized_variant_temp[j] == 1)) {
                                                        S[0].resetSolver(feature_count);
                                                        S[0].resetSimpSolver(feature_count);

                                                        S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                        S[0].eliminate(true);

                                                        if (!S[0].okay()) {
                                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                            exit(0);
                                                        }

                                                        vec<Lit> dummy1;

                                                        dummy1.push(~mkLit(i + 1));
                                                        dummy1.push(mkLit(j));

                                                        lbool ret1 = S[0].solveLimited(dummy1);
                                                        if (ret1 == l_True) {
                                                            for (int k = 0; k < feature_count; k++) {
                                                                if (S[0].model[k] != l_Undef) {
                                                                    if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                    else optimized_variant[k] = 0;
                                                                }
                                                                else {
                                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                                    exit(0);
                                                                }
                                                            }

                                                            start_point_x = x;
                                                            start_point_j = j;

                                                            found_valid = true;
                                                            break;
                                                        }
                                                        else {
                                                            invalid_inter_off_on = 1;

                                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                            invalids_uncovered[i]++;

                                                            invalids_total++;

                                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                                        }
                                                    }

                                                    if (invalid_inter_off_off == 0 && inter_off_off == 1 && !(optimized_variant_temp[i + 1] == 0 && optimized_variant_temp[j] == 0)) {
                                                        S[0].resetSolver(feature_count);
                                                        S[0].resetSimpSolver(feature_count);

                                                        S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                        S[0].eliminate(true);

                                                        if (!S[0].okay()) {
                                                            printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                            exit(0);
                                                        }

                                                        vec<Lit> dummy1;

                                                        dummy1.push(~mkLit(i + 1));
                                                        dummy1.push(~mkLit(j));

                                                        lbool ret1 = S[0].solveLimited(dummy1);
                                                        if (ret1 == l_True) {
                                                            for (int k = 0; k < feature_count; k++) {
                                                                if (S[0].model[k] != l_Undef) {
                                                                    if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                    else optimized_variant[k] = 0;
                                                                }
                                                                else {
                                                                    printf("UNDEFINED VARIABLES, exiting...\n");
                                                                    exit(0);
                                                                }
                                                            }

                                                            start_point_x = x;
                                                            start_point_j = j;

                                                            found_valid = true;
                                                            break;
                                                        }
                                                        else {
                                                            invalid_inter_off_off = 1;

                                                            invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                            invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                            invalids_uncovered[i]++;

                                                            invalids_total++;

                                                            if (invalids_uncovered[i] == uncovered[i]) break;
                                                        }
                                                    }

                                                }
                                            }
                                            else skip = false;
                                        }
                                    }
                                    if (found_valid) break;
                                }

                                if (!found_valid) {
                                    cout << "End reached, exiting..." << endl;
                                    break;
                                }

                                hipMemcpy(optimized_variant_c, optimized_variant, feature_count * sizeof(char), hipMemcpyHostToDevice);

                                reset_M_values << < numBlocksFeatures, blockSize >> > (feature_count, M_values_c);
                                calculate_init_M_values << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interactions_c, optimized_variant_c, M_values_c);

                                calculate_clause_values << < numBlocksClauses, blockSize >> > (reduced_clause_count, current_clause_values_c, min_clause_values_c, clause_list_c, lits_in_clause_limit, clause_list_sizes_c, optimized_variant_c);
                            }
                            else break;

                        }

                    }
                    else {
                        int gain_value;

                        if (!disable_qip) {
                            current_step_m[0] = it_count % step_size;

                            hipMemcpy(current_step_m_c, current_step_m, sizeof(int), hipMemcpyHostToDevice);

                            reset_dual_flip_counts << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, is_flip_valid_c, dual_invalids_c, dual_valids_c, valid_thresholds_c);

                            calculate_dual_flip_validity_optimized << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, total_lits_in_file, current_clause_values_c, min_clause_values_c, feat_to_clause_reference_c, clause_to_feat_reference_c, clause_list_c, clause_list_sizes_c, lits_in_clause_limit, optimized_variant_c, is_flip_valid_c, dual_invalids_c, dual_valids_c, valid_thresholds_c);

                            calculate_max_gain_dual_with_validity_check_2 << < numBlocksFeatures, blockSize >> > (feature_count, feature_interactions_c, optimized_variant_c, M_values_c, max_indices_j_m_c, max_gains_m_c, step_size, current_step_m_c, is_flip_valid_c, dual_invalids_c, dual_valids_c, valid_thresholds_c, are_feature_pairs_dependent_c);

                            hipMemcpy(max_gains_m, max_gains_m_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);
                            hipMemcpy(max_indices_j_m, max_indices_j_m_c, feature_count * sizeof(int), hipMemcpyDeviceToHost);

                            gain_value = 0;

                            for (int i = 0; i < feature_count; i++) {
                                if (max_gains_m[i] > gain_value) {
                                    gain_value = max_gains_m[i];
                                    max_index_i_m[0] = i;
                                    max_index_j_m[0] = max_indices_j_m[i];
                                }
                            }
                        }
                        else gain_value = 0;

                        if (gain_value > 0) {
                            hipMemcpy(max_index_i_m_c, max_index_i_m, sizeof(int), hipMemcpyHostToDevice);
                            hipMemcpy(max_index_j_m_c, max_index_j_m, sizeof(int), hipMemcpyHostToDevice);

                            adapt_M_values_dual << < numBlocksFeatures, blockSize >> > (feature_count, feature_interactions_c, optimized_variant_c, M_values_c, max_index_i_m_c, max_index_j_m_c);

                            int numBlocksAffectedClauses = int(round(static_cast<double>(clauses_with_lit[max_index_i_m[0]]) + blockSize - 1) / static_cast<double>(blockSize));

                            adapt_clause_values << < numBlocksAffectedClauses, blockSize >> > (current_clause_values_c, clauses_with_lit_c, feat_to_clause_reference_c, reference_start_points_c, optimized_variant_c, max_index_i_m_c);

                            if (max_index_j_m[0] != -1) {
                                numBlocksAffectedClauses = int(round(static_cast<double>(clauses_with_lit[max_index_j_m[0]]) + blockSize - 1) / static_cast<double>(blockSize));
                                adapt_clause_values << < numBlocksAffectedClauses, blockSize >> > (current_clause_values_c, clauses_with_lit_c, feat_to_clause_reference_c, reference_start_points_c, optimized_variant_c, max_index_j_m_c);
                            }

                            optimize_variant_dual << < 1, 1 >> > (optimized_variant_c, max_index_i_m_c, max_index_j_m_c);

                            if (run_counter == 0) total_gain += gain_value;

                            zero_counter = 0;
                        }
                        else {
                            zero_counter++;

                            if (zero_counter == step_size) {

                                if (total_gain > 0) break;

                                if (run_counter == 0) {
                                    hipMemcpy(optimized_variant, optimized_variant_c, feature_count * sizeof(char), hipMemcpyDeviceToHost);
                                    hipMemcpy(feature_interactions, feature_interactions_c, feature_interaction_count * sizeof(char), hipMemcpyDeviceToHost);

                                    for (int i = 0; i < feature_count; i++) optimized_variant_temp[i] = optimized_variant[i];
                                }

                                if (run_counter < retries_per_optimization) {

                                    if (run_counter >= 1) {
                                        reset_novel_fi_count << < 1, 1 >> > (novel_fi_count_c);
                                        calculate_sample_gain_optimized << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, optimized_variant_c, novel_fi_count_c, feature_interactions_c);
                                        hipMemcpy(novel_fi_count, novel_fi_count_c, sizeof(unsigned long long), hipMemcpyDeviceToHost);

                                        if (novel_fi_count[0] > uncovered_without - uncovered_total + total_gain) {
                                            cout << "Found something better!" << endl;
                                            use_alternative = true;
                                            break;
                                        }
                                    }

                                    zero_counter = 0;
                                    run_counter++;

                                    bool skip = false;

                                    if (start_point_x != 0 || start_point_j != 0) skip = true;

                                    bool found_valid = false;
                                    unsigned long long inter_index;
                                    char inter_off_off, inter_off_on, inter_on_off, inter_on_on;
                                    char invalid_inter_off_off, invalid_inter_off_on, invalid_inter_on_off, invalid_inter_on_on;
                                    char inter_bin_encoding, invalid_inter_bin_encoding;

                                    for (int x = start_point_x; x < feature_count - 1; x++) {

                                        int i = shuffled_features[x];

                                        if (invalids_uncovered[i] < uncovered[i]) {
                                            for (int j = start_point_j; j < i + 1; j++) {

                                                if (!skip) {
                                                    inter_index = (i * (i + 1)) / 2 + j;

                                                    inter_bin_encoding = feature_interactions[inter_index];
                                                    invalid_inter_bin_encoding = invalid_feature_interactions[inter_index];

                                                    if (inter_bin_encoding != 0 && inter_bin_encoding != invalid_inter_bin_encoding) {

                                                        if (inter_bin_encoding >= 8) {
                                                            inter_on_on = 1;
                                                            inter_bin_encoding -= 8;
                                                        }
                                                        else {
                                                            inter_on_on = 0;
                                                        }

                                                        if (inter_bin_encoding >= 4) {
                                                            inter_on_off = 1;
                                                            inter_bin_encoding -= 4;
                                                        }
                                                        else {
                                                            inter_on_off = 0;
                                                        }

                                                        if (inter_bin_encoding >= 2) {
                                                            inter_off_on = 1;
                                                            inter_bin_encoding -= 2;
                                                        }
                                                        else {
                                                            inter_off_on = 0;
                                                        }

                                                        if (inter_bin_encoding == 1) inter_off_off = 1;
                                                        else inter_off_off = 0;

                                                        if (invalid_inter_bin_encoding >= 8) {
                                                            invalid_inter_on_on = 1;
                                                            invalid_inter_bin_encoding -= 8;
                                                        }
                                                        else {
                                                            invalid_inter_on_on = 0;
                                                        }

                                                        if (invalid_inter_bin_encoding >= 4) {
                                                            invalid_inter_on_off = 1;
                                                            invalid_inter_bin_encoding -= 4;
                                                        }
                                                        else {
                                                            invalid_inter_on_off = 0;
                                                        }

                                                        if (invalid_inter_bin_encoding >= 2) {
                                                            invalid_inter_off_on = 1;
                                                            invalid_inter_bin_encoding -= 2;
                                                        }
                                                        else {
                                                            invalid_inter_off_on = 0;
                                                        }

                                                        if (invalid_inter_bin_encoding == 1) invalid_inter_off_off = 1;
                                                        else invalid_inter_off_off = 0;

                                                        if (invalid_inter_on_on == 0 && inter_on_on == 1 && !(optimized_variant_temp[i + 1] == 1 && optimized_variant_temp[j] == 1)) {
                                                            S[0].resetSolver(feature_count);
                                                            S[0].resetSimpSolver(feature_count);

                                                            S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                            S[0].eliminate(true);

                                                            if (!S[0].okay()) {
                                                                printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                                exit(0);
                                                            }

                                                            vec<Lit> dummy1;

                                                            dummy1.push(mkLit(i + 1));
                                                            dummy1.push(mkLit(j));

                                                            lbool ret1 = S[0].solveLimited(dummy1);

                                                            if (ret1 == l_True) {
                                                                for (int k = 0; k < feature_count; k++) {
                                                                    if (S[0].model[k] != l_Undef) {
                                                                        if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                        else optimized_variant[k] = 0;
                                                                    }
                                                                    else {
                                                                        printf("UNDEFINED VARIABLES, exiting...\n");
                                                                        exit(0);
                                                                    }
                                                                }

                                                                start_point_x = x;
                                                                start_point_j = j;

                                                                found_valid = true;
                                                                break;
                                                            }
                                                            else {
                                                                invalid_inter_on_on = 1;

                                                                invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                                invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                                invalids_uncovered[i]++;

                                                                invalids_total++;

                                                                if (invalids_uncovered[i] == uncovered[i]) break;
                                                            }
                                                        }

                                                        if (invalid_inter_on_off == 0 && inter_on_off == 1 && !(optimized_variant_temp[i + 1] == 1 && optimized_variant_temp[j] == 0)) {
                                                            S[0].resetSolver(feature_count);
                                                            S[0].resetSimpSolver(feature_count);

                                                            S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                            S[0].eliminate(true);

                                                            if (!S[0].okay()) {
                                                                printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                                exit(0);
                                                            }

                                                            vec<Lit> dummy1;

                                                            dummy1.push(mkLit(i + 1));
                                                            dummy1.push(~mkLit(j));

                                                            lbool ret1 = S[0].solveLimited(dummy1);
                                                            if (ret1 == l_True) {
                                                                for (int k = 0; k < feature_count; k++) {
                                                                    if (S[0].model[k] != l_Undef) {
                                                                        if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                        else optimized_variant[k] = 0;
                                                                    }
                                                                    else {
                                                                        printf("UNDEFINED VARIABLES, exiting...\n");
                                                                        exit(0);
                                                                    }
                                                                }

                                                                start_point_x = x;
                                                                start_point_j = j;

                                                                found_valid = true;
                                                                break;
                                                            }
                                                            else {
                                                                invalid_inter_on_off = 1;

                                                                invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                                invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                                invalids_uncovered[i]++;

                                                                invalids_total++;

                                                                if (invalids_uncovered[i] == uncovered[i]) break;
                                                            }
                                                        }

                                                        if (invalid_inter_off_on == 0 && inter_off_on == 1 && !(optimized_variant_temp[i + 1] == 0 && optimized_variant_temp[j] == 1)) {
                                                            S[0].resetSolver(feature_count);
                                                            S[0].resetSimpSolver(feature_count);

                                                            S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                            S[0].eliminate(true);

                                                            if (!S[0].okay()) {
                                                                printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                                exit(0);
                                                            }

                                                            vec<Lit> dummy1;

                                                            dummy1.push(~mkLit(i + 1));
                                                            dummy1.push(mkLit(j));

                                                            lbool ret1 = S[0].solveLimited(dummy1);
                                                            if (ret1 == l_True) {
                                                                for (int k = 0; k < feature_count; k++) {
                                                                    if (S[0].model[k] != l_Undef) {
                                                                        if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                        else optimized_variant[k] = 0;
                                                                    }
                                                                    else {
                                                                        printf("UNDEFINED VARIABLES, exiting...\n");
                                                                        exit(0);
                                                                    }
                                                                }

                                                                start_point_x = x;
                                                                start_point_j = j;

                                                                found_valid = true;
                                                                break;
                                                            }
                                                            else {
                                                                invalid_inter_off_on = 1;

                                                                invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                                invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                                invalids_uncovered[i]++;

                                                                invalids_total++;

                                                                if (invalids_uncovered[i] == uncovered[i]) break;
                                                            }
                                                        }

                                                        if (invalid_inter_off_off == 0 && inter_off_off == 1 && !(optimized_variant_temp[i + 1] == 0 && optimized_variant_temp[j] == 0)) {
                                                            S[0].resetSolver(feature_count);
                                                            S[0].resetSimpSolver(feature_count);

                                                            S[0].setTargetValue(0, feature_count, optimized_variant_temp);

                                                            S[0].eliminate(true);

                                                            if (!S[0].okay()) {
                                                                printf("Solved by simplification - UNSATISFIABLE, exiting...\n");
                                                                exit(0);
                                                            }

                                                            vec<Lit> dummy1;

                                                            dummy1.push(~mkLit(i + 1));
                                                            dummy1.push(~mkLit(j));

                                                            lbool ret1 = S[0].solveLimited(dummy1);
                                                            if (ret1 == l_True) {
                                                                for (int k = 0; k < feature_count; k++) {
                                                                    if (S[0].model[k] != l_Undef) {
                                                                        if (S[0].model[k] == l_True) optimized_variant[k] = 1;
                                                                        else optimized_variant[k] = 0;
                                                                    }
                                                                    else {
                                                                        printf("UNDEFINED VARIABLES, exiting...\n");
                                                                        exit(0);
                                                                    }
                                                                }

                                                                start_point_x = x;
                                                                start_point_j = j;

                                                                found_valid = true;
                                                                break;
                                                            }
                                                            else {
                                                                invalid_inter_off_off = 1;

                                                                invalid_inter_bin_encoding = invalid_inter_off_off + 2 * invalid_inter_off_on + 4 * invalid_inter_on_off + 8 * invalid_inter_on_on;

                                                                invalid_feature_interactions[inter_index] = invalid_inter_bin_encoding;

                                                                invalids_uncovered[i]++;

                                                                invalids_total++;

                                                                if (invalids_uncovered[i] == uncovered[i]) break;
                                                            }
                                                        }

                                                    }
                                                }
                                                else skip = false;
                                            }
                                        }
                                        if (found_valid) break;
                                    }

                                    if (!found_valid) {
                                        cout << "End reached, exiting..." << endl;
                                        break;
                                    }

                                    hipMemcpy(optimized_variant_c, optimized_variant, feature_count * sizeof(char), hipMemcpyHostToDevice);

                                    reset_M_values << < numBlocksFeatures, blockSize >> > (feature_count, M_values_c);
                                    calculate_init_M_values << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interactions_c, optimized_variant_c, M_values_c);

                                    calculate_clause_values << < numBlocksClauses, blockSize >> > (reduced_clause_count, current_clause_values_c, min_clause_values_c, clause_list_c, lits_in_clause_limit, clause_list_sizes_c, optimized_variant_c);
                                }
                                else break;
                            }
                        }

                    }

                    it_count++;
                }

                if (!use_alternative) {
                    if (run_counter >= 1) {
                        reset_novel_fi_count << < 1, 1 >> > (novel_fi_count_c);
                        calculate_sample_gain_optimized << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, optimized_variant_c, novel_fi_count_c, feature_interactions_c);
                        hipMemcpy(novel_fi_count, novel_fi_count_c, sizeof(unsigned long long), hipMemcpyDeviceToHost);

                        if (novel_fi_count[0] > uncovered_without - uncovered_total + total_gain) {
                            cout << "Found something better!" << endl;
                            use_alternative = true;
                        }
                    }
                }

                unsigned uncovered_total_before = uncovered_total;

                if (!use_alternative) {
                    if (total_gain == 0) {
                        for (int i = 0; i < feature_count; i++) optimized_variant[i] = optimized_variant_temp[i];
                        hipMemcpy(optimized_variant_c, optimized_variant, feature_count * sizeof(char), hipMemcpyHostToDevice);
                    }

                    uncovered_total -= total_gain;
                }
                else {
                    uncovered_total = uncovered_without - novel_fi_count[0];
                }

                if (uncovered_total_before > uncovered_total) {
                    not_zero_counter++;
                }

                /*
                if (uncovered_total_before > uncovered_total) {
                    hipMemcpy(optimized_variant, optimized_variant_c, feature_count * sizeof(char), hipMemcpyDeviceToHost);

                    ifstream file1(cnf_file_name);

                    bool found;
                    int sat_counter = 0;
                    int unsat_counter = 0;
                    int number;
                    bool negative, comment, command;

                    if (file1.is_open()) {
                        while (getline(file1, file_line)) {
                            number = 0;
                            negative = false;
                            found = false;

                            comment = false;
                            command = false;

                            char_counter = 0;

                            for (char& c : file_line) {
                                int index = int(c);

                                if (char_counter == 0 && index == 99) {
                                    comment = true;
                                    break;
                                }

                                if (char_counter == 0 && index == 112) {
                                    command = true;
                                    break;
                                }

                                if (index >= 48 && index <= 57) {
                                    if (number == 0) {
                                        number = index - 48;
                                    }
                                    else {
                                        number *= 10;
                                        number += index - 48;
                                    }
                                }
                                else if (index == 45) {
                                    negative = true;
                                }
                                else {
                                    if (number > 0) {
                                        if (negative) {
                                            if (optimized_variant[number - 1] == 0) found = true;
                                        }
                                        else {
                                            if (optimized_variant[number - 1] == 1) found = true;
                                        }
                                    }

                                    number = 0;
                                    negative = false;
                                }

                                char_counter++;
                            }

                            if (comment || command) continue;

                            if (number > 0) {
                                if (negative) {
                                    if (optimized_variant[number - 1] == 0) found = true;
                                }
                                else {
                                    if (optimized_variant[number - 1] == 1) found = true;
                                }
                            }

                            if (!found) unsat_counter++;
                            else sat_counter++;
                        }

                        file1.close();
                    }

                    std::cout << "\nCLAUSES SAT / UNSAT: " << sat_counter << " / " << unsat_counter << std::endl;
                    if (unsat_counter != 0) cout << "!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!" << endl;
                }
                */

                write_optimized_to_current_sample << < numBlocksFeatures, blockSize >> > (feature_count, optimized_variant_c, skip_sample_index_c, current_sample_c);

                if (modify_oldest) {
                    variant_ages[skip_sample_index[0]] = age_counter;
                    age_counter++;
                }

                append_interactions << < int(round(static_cast<double>(parallel_tasks) + blockSize - 1) / static_cast<double>(blockSize)), blockSize >> > (parallel_tasks, feature_count, feature_interaction_count, optimized_variant_c, feature_interactions_c, uncovered_c);

                cout << "Variant: " << skip_sample_index[0] << " - newly covered interactions: " << uncovered_total_before - uncovered_total << " - iterations: " << it_count + 1 << endl;
            }


            if (not_zero_counter > 2) retries_per_optimization++;
            else retries_per_optimization--;

            if (retries_per_optimization > max_retries_per_optimization) retries_per_optimization = max_retries_per_optimization;
            if (retries_per_optimization < min_retries_per_optimization) retries_per_optimization = min_retries_per_optimization;

        }

        cout << "===============================" << endl;

    }

    hipMemcpy(uncovered, uncovered_c, (feature_count - 1) * sizeof(int), hipMemcpyDeviceToHost);

    unsigned long long uncovered_total = 0;

    for (int i = 0; i < feature_count - 1; i++) {
        uncovered_total += uncovered[i];
    }

    cout << ">>>>>> Uncovered interactions: " << uncovered_total << " <<<<<<" << endl;

    uncovered_fi.push_back(uncovered_total);
    invalid_fi.push_back(invalids_total);
    variant_id.push_back(variant_counter);

    auto elapsed1 = std::chrono::high_resolution_clock::now() - start1;

    long long microseconds1 = std::chrono::duration_cast<std::chrono::microseconds>(
        elapsed1).count();

    cout << "On GPU duration (us, ms, s): " << microseconds1 << ", " << microseconds1 / 1000 << ", " << microseconds1 / 1000000 << endl;

    cout << "=======================================" << endl;


    for (int i = 0; i < uncovered_fi.size(); i++) {
        cout << uncovered_fi[i] << ",";
    }
    cout << endl;

    for (int i = 0; i < uncovered_fi.size(); i++) {
        cout << invalid_fi[i] << ",";
    }
    cout << endl;

    for (int i = 0; i < uncovered_fi.size(); i++) {
        cout << variant_id[i] << ",";
    }
    cout << endl;

    /*
    char invalid_inter_bin_encoding;
    char invalid_inter_off_off, invalid_inter_off_on, invalid_inter_on_off, invalid_inter_on_on;

    std::ofstream file("C:\\Users\\lenna\\Downloads\\invalid_interactions\\erp_system.csv");

    if (!file.is_open()) {
        std::cerr << "Failed to open file!" << std::endl;
        return 1;
    }

    std::vector<std::vector<std::string>> csv_data;


    for (int i = 1; i < feature_count; i++) {

        for (int j = 0; j < i; j++) {

            inter_index = ((i - 1) * i) / 2 + j;

            invalid_inter_bin_encoding = invalid_feature_interactions[inter_index];

            if (inter_bin_encoding == 1) inter_off_off = 1;
            else inter_off_off = 0;

            if (invalid_inter_bin_encoding >= 8) {
                invalid_inter_on_on = 1;
                invalid_inter_bin_encoding -= 8;
            }
            else {
                invalid_inter_on_on = 0;
            }

            if (invalid_inter_bin_encoding >= 4) {
                invalid_inter_on_off = 1;
                invalid_inter_bin_encoding -= 4;
            }
            else {
                invalid_inter_on_off = 0;
            }

            if (invalid_inter_bin_encoding >= 2) {
                invalid_inter_off_on = 1;
                invalid_inter_bin_encoding -= 2;
            }
            else {
                invalid_inter_off_on = 0;
            }

            if (invalid_inter_bin_encoding == 1) invalid_inter_off_off = 1;
            else invalid_inter_off_off = 0;

            if (invalid_inter_on_on == 1) csv_data.push_back({ "" + std::to_string(i+1), "" + std::to_string(j+1)});
            if (invalid_inter_on_off == 1) csv_data.push_back({ "" + std::to_string(i + 1), "-" + std::to_string(j + 1) });
            if (invalid_inter_off_on == 1) csv_data.push_back({ "-" + std::to_string(i + 1), "" + std::to_string(j + 1) });
            if (invalid_inter_off_off == 1) csv_data.push_back({ "-" + std::to_string(i + 1), "-" + std::to_string(j + 1) });
        }
    }

    for (const auto& row : csv_data) {
        for (size_t i = 0; i < row.size(); ++i) {
            file << row[i];
            if (i != row.size() - 1) file << ",";
        }
        file << "\n";
    }

    file.close();
    std::cout << "CSV file created successfully." << std::endl;
    */

    hipFree(clause_list_c);
    hipFree(clause_list_sizes_c);
    hipFree(current_clause_values_c);
    hipFree(min_clause_values_c);
    hipFree(feat_to_clause_reference_c);
    hipFree(clauses_with_lit_c);
    hipFree(reference_start_points_c);
    hipFree(is_flip_valid_c);

    hipFree(current_sample_size_c);
    hipFree(current_sample_c);
    hipFree(skip_sample_index_c);
    hipFree(sampled_variants_c);
    hipFree(novel_fi_counts_c);
    hipFree(novel_fi_count_c);
    hipFree(feature_interactions_c);
    hipFree(one_counts_c);
    hipFree(max_index_c);
    hipFree(max_index_m_c);

    hipFree(max_indices_j_m_c);
    hipFree(max_gains_m_c);
    hipFree(max_index_i_m_c);
    hipFree(max_index_j_m_c);
    hipFree(current_step_m_c);

    hipFree(M_values_c);
    hipFree(optimized_variant_c);
    hipFree(uncovered_c);

    cout << "\nCUDA cleanup completed..." << endl;

    free(invalid_feature_interactions);
    free(invalids_uncovered);
    free(active_literals);
    free(inactive_literals);
    free(clause_list);
    free(clause_list_sizes);
    free(feat_to_clause_reference);
    free(clauses_with_lit);
    free(current_reference_iterators);
    free(reference_start_points);
    free(is_flip_valid);

    free(current_sample_size);
    free(current_sample);
    free(skip_sample_index);
    free(sampled_variants);
    free(novel_fi_counts);
    free(feature_interactions);
    free(max_index);
    free(max_index_m);
    free(max_gain_m);

    free(max_indices_j_m);
    free(max_gains_m);
    free(max_index_i_m);
    free(max_index_j_m);
    free(current_step_m);

    free(M_values);
    free(optimized_variant);
    free(optimized_variant_temp);
    free(uncovered);

    cout << "Host cleanup completed..." << endl;
}